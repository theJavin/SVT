#include "hip/hip_runtime.h"
// nvcc SVT4.0.cu -o svt4.0 -lglut -lm -lGLU -lGL
//To stop hit "control c" in the window you launched it from. stuff

// Length will be in millimeters
// Time will be in milliseconds
// Mass will be in ???

// Fiber length 100 micrometers or 0.1 millimeters
// Sodium wave speed .5 meters/sec or 0.5 millimeters/millisec

#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define PI 3.141592654

#define STOP_TIME 60000.0
#define DT  0.001

//#define NumberOfNodes 266 //266 //62
//#define NumberOfMuscles 552 //552 //132
//#define LinksPerNode 24 //24

// Globals
int DrawRate;
int Pause;

int NumberOfNodes;
int NumberOfMuscles;
int LinksPerNode;

float4 *NodePosition, *NodeVelocity, *NodeForce;
float *NodeMass;
int *NodeLinks; // The nodes that this node is connected to
int *NodeMuscles; // The muscle that connects this node to ther other nodes
int *NodeAblatedYesNo;

float MassOfAtria;

// How the muscle will act without contraction.
int *MuscleConectionA, *MuscleConectionB;
float *MuscleMass, *MuscleLength, *MuscleRelaxedStrength;
float *MuscleCompresionStopFraction;  // 0.7 is the standard value
float3 *MuscleColor;
float MuscleCompresionMultiplier;
float MuscleTentionMultiplier;
float Viscosity;

// Muscle contraction parameters
int *ContractionOnOff;
float *ContractionTimer;
float *ActionPotentialSpeed; //0.5 is a good value.
float *ActionPotentialDuration;
float *ContractionDuration; // 100.0 is a good value
float *RelaxationDuration; // 200.0 is a good value
float *ContractionStrength; // 5.0 is a good value

float BloodPresure;
float BloodPresureScaling;
float BeatPeriod;

int   DrawTimer; 
float RunTime;
float BeatTimer;
float4 CenterOfSimulation;

// Window globals
static int Window;
int XWindowSize;
int YWindowSize; 
double Near;
double Far;
double EyeX;
double EyeY;
double EyeZ;
double CenterX;
double CenterY;
double CenterZ;
double UpX;
double UpY;
double UpZ;

// Prototyping functions
void allocateMemory(int, int);
void setNodesAndMusclesCircle(int); 
void setNodesAndMusclesSphere(int);
void linkNodesToMuscles();
void setMuscleAttributesAndNodeMasses(int);
void drawPicture();
void generalMuscleForces();
void outwardPresure();
void turnOnNodeMuscles(int);
int contractionForces(float, float);
void dampingForce();
void moveNodes(float, float);
void hardCodedAblatedNodes();
void hardCodedEctopicEvents(float, float);
void n_body(float);
void setup();
void KeyPressed(unsigned char, int, int);
void mymouse(int, int, int, int);
void Display(void);
void reshape(int, int);
void allocateMemory(int, int);

#include "./setNodesAndMuscles.h"
#include "./callBackFunctions.h"

void allocateMemory(int type, int divisions)
{
	if(type == 1) // Circle
	{
		NumberOfNodes = divisions;
		NumberOfMuscles = divisions;
		LinksPerNode = 2;
	}
	else if(type == 2) //Sphere
	{
		NumberOfNodes = divisions*(divisions/2 - 1) + 2;
		NumberOfMuscles = divisions + (divisions*2)*(divisions/2 - 1);
		LinksPerNode = divisions;
	}
	else if(type == 3) //Sphere with thickness
	{
		printf("\n Thick spheres are not it yet.\n");
		exit(0);
	}
	else
	{
		printf("\n Bad object type.\n");
		exit(0);
	}
	
	NodePosition = (float4*)malloc(NumberOfNodes*sizeof(float4));
	NodeVelocity = (float4*)malloc(NumberOfNodes*sizeof(float4));
	NodeForce    = (float4*)malloc(NumberOfNodes*sizeof(float4));
	
	NodeMass = (float*)malloc(NumberOfNodes*sizeof(float));
	NodeLinks = (int*)malloc(NumberOfNodes*LinksPerNode*sizeof(int));
	NodeMuscles = (int*)malloc(NumberOfNodes*LinksPerNode*sizeof(int));
	NodeAblatedYesNo = (int*)malloc(NumberOfNodes*sizeof(int));

	// How the muscle will act without contraction.
	MuscleConectionA = (int*)malloc(NumberOfMuscles*sizeof(int));
	MuscleConectionB = (int*)malloc(NumberOfMuscles*sizeof(int));
	MuscleMass = (float*)malloc(NumberOfMuscles*sizeof(float));
	MuscleLength = (float*)malloc(NumberOfMuscles*sizeof(float));
	MuscleRelaxedStrength = (float*)malloc(NumberOfMuscles*sizeof(float));
	MuscleCompresionStopFraction = (float*)malloc(NumberOfMuscles*sizeof(float));
	MuscleColor = (float3*)malloc(NumberOfMuscles*sizeof(float3));


	// Muscle contraction parameters
	ContractionOnOff = (int*)malloc(NumberOfMuscles*sizeof(int));
	ContractionTimer = (float*)malloc(NumberOfMuscles*sizeof(float));
	ActionPotentialSpeed = (float*)malloc(NumberOfMuscles*sizeof(float));
	ActionPotentialDuration = (float*)malloc(NumberOfMuscles*sizeof(float));
	ContractionDuration = (float*)malloc(NumberOfMuscles*sizeof(float));
	RelaxationDuration = (float*)malloc(NumberOfMuscles*sizeof(float));
	ContractionStrength = (float*)malloc(NumberOfMuscles*sizeof(float));
}

void linkNodesToMuscles()
{
	//Setting the ends of the muscles
	int index = 0;
	for(int i = 0; i < NumberOfNodes; i++)
	{
		for(int j = 0; j < LinksPerNode; j++)
		{
			if(NodeLinks[i*LinksPerNode + j] != -1)
			{
				if(i < NodeLinks[i*LinksPerNode + j])
				{
					if(NumberOfMuscles <= index)
					{
						printf("\nTSU Error: number of muscles is out of bounds\n");
						exit(0);
					} 
					MuscleConectionA[index] = i;
					MuscleConectionB[index] = NodeLinks[i*LinksPerNode + j];
					index++;
				}
			}
		}
	}
	
	// Setting the node muscles. Each node will have a list of nodes they are attached to (NodeLinks[][]) and the muscle that attaches it to that node (NodeMuscles[][]).
	// Setting them all to -1 first.
	for(int i = 0; i < NumberOfNodes; i++)
	{
		for(int j = 0; j < LinksPerNode; j++)
		{
			NodeMuscles[i*LinksPerNode + j] = -1;
		}	
	}
	for(int i = 0; i < NumberOfNodes; i++)
	{
		for(int j = 0; j < LinksPerNode; j++)
		{
			if(NodeLinks[i*LinksPerNode + j] != -1)
			{
				for(int k = 0; k < NumberOfMuscles; k++)
				{
					if((MuscleConectionA[k] == i && MuscleConectionB[k] == NodeLinks[i*LinksPerNode + j]) || (MuscleConectionA[k] == NodeLinks[i*LinksPerNode + j] && MuscleConectionB[k] == i))
					{
						NodeMuscles[i*LinksPerNode + j] = k;
					}
				}
			}
		}
	}
}

void setMuscleAttributesAndNodeMasses(int divisions)
{	
	float dx, dy, dz, d;
	float sum, totalLengthOfAllMuscles;
	
	MassOfAtria = 1.0;
	MuscleCompresionMultiplier = 50.0;
	MuscleTentionMultiplier = 50.0;
	Viscosity = 1.0/divisions;
	BloodPresureScaling = divisions*(divisions/2 - 1) + 2;  // Set it to the number of nodes for a sphere in both the circle and the sphere simulation. Think about this some more. Hard to have presure in 1D.
	BloodPresure = 1.0/BloodPresureScaling;
	CenterOfSimulation.x = 0.0;
	CenterOfSimulation.y = 0.0;
	CenterOfSimulation.z = 0.0;
	
	//Finding the length of each muscle and the total length of all muscles.
	totalLengthOfAllMuscles = 0.0;
	for(int i = 0; i < NumberOfMuscles; i++)
	{	
		dx = NodePosition[MuscleConectionA[i]].x - NodePosition[MuscleConectionB[i]].x;
		dy = NodePosition[MuscleConectionA[i]].y - NodePosition[MuscleConectionB[i]].y;
		dz = NodePosition[MuscleConectionA[i]].z - NodePosition[MuscleConectionB[i]].z;
		d = sqrt(dx*dx + dy*dy + dz*dz);
		MuscleLength[i] = d;
		totalLengthOfAllMuscles += d;
	}
	
	// Setting the mass of all muscles.
	for(int i = 0; i < NumberOfMuscles; i++)
	{	
		MuscleMass[i] = (MuscleLength[i]/totalLengthOfAllMuscles)*MassOfAtria;
	}
	
	// Setting other parameters
	for(int i = 0; i < NumberOfMuscles; i++)
	{	
		MuscleRelaxedStrength[i] = 0.1*MuscleLength[i];
		MuscleCompresionStopFraction[i] = 0.7; // should be 0.7
		ContractionOnOff[i] = 0;
		ContractionTimer[i] = 0.0;
		ActionPotentialSpeed[i] = 0.2; // 0.2
		ActionPotentialDuration[i] = MuscleLength[i]/ActionPotentialSpeed[i];
		ContractionDuration[i] = 20.0;  // 100.0
		RelaxationDuration[i] = 40.0;  // 200.0
		ContractionStrength[i] = 0.2*MuscleLength[i]; //0.1;
		MuscleColor[i].x = 1.0;
		MuscleColor[i].y = 0.0;
		MuscleColor[i].z = 0.0;
		NodeAblatedYesNo[i] = 0; // Setting all nodes to not ablated.
	}
	
	// Setting the node masses
	for(int i = 0; i < NumberOfNodes; i++)
	{
		sum = 0.0;
		for(int j = 0; j < LinksPerNode; j++)
		{
			if(NodeMuscles[i*LinksPerNode + j] != -1)
			{
				sum += MuscleMass[NodeMuscles[i*LinksPerNode + j]];
			}
		}
		NodeMass[i] = sum/2.0;
	}
}

void hardCodedAblatedNodes()
{
	//Nodes to ablate
	for(int i = 1; i < 23; i++)
	{	
		//NodeAblatedYesNo[i] = 1;
	}
	
	//NodeAblatedYesNo[49] = 1;
}

void hardCodedEctopicEvents(float time, float dt)
{
	float er = dt/2.0;
	
	if((50.0 - er <= time) && (time < 220.0 + er))
	{
		//turnOnNodeMuscles(31);
	}
	
	if((51.0 - er <= time) && (time < 230.0 + er))
	{
		//turnOnNodeMuscles(41);
	}
	
	if((240.0 - er <= time) && (time < 230.0 + er))
	{
		//turnOnNodeMuscles(59);
	}
}

void drawPicture()
{
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	
	glColor3d(1.0,1.0,0.0);
	glPushMatrix();
	glTranslatef(NodePosition[0].x, NodePosition[0].y, NodePosition[0].z);
	glutSolidSphere(0.03,20,20);
	glPopMatrix();
	
	// Drawing nodes
	for(int i = 1; i < NumberOfNodes; i++)
	{
		if(NodeAblatedYesNo[i] == 0)
		{
			glColor3d(0.0,1.0,0.0);
		}
		else
		{
			glColor3d(1.0,1.0,1.0);
		}
		glPushMatrix();
		glTranslatef(NodePosition[i].x, NodePosition[i].y, NodePosition[i].z);
		glutSolidSphere(0.01,20,20);
		glPopMatrix();	
	}
	
	/*
	glColor3d(1.0,1.0,0.0);
	glPushMatrix();
	glTranslatef(NodePosition[NumberOfNodes-1].x, NodePosition[NumberOfNodes-1].y, NodePosition[NumberOfNodes-1].z);
	glutSolidSphere(0.03,20,20);
	glPopMatrix();
	*/
	
	// Drawing muscles
	glColor3d(1.0,0.0,0.0);
	for(int i = 0; i < NumberOfNodes; i++)
	{
		for(int j = 0; j < LinksPerNode; j++)
		{
			if(NodeLinks[i*LinksPerNode + j] != -1)
			{
				glColor3d(MuscleColor[NodeMuscles[i*LinksPerNode + j]].x, MuscleColor[NodeMuscles[i*LinksPerNode + j]].y, MuscleColor[NodeMuscles[i*LinksPerNode + j]].z);
				//glLineWidth(1.0/(Px[i]-Px[NodeLinks[i*LinksPerNode + j]]));
				glBegin(GL_LINES);
					glVertex3f(NodePosition[i].x, NodePosition[i].y, NodePosition[i].z);
					glVertex3f(NodePosition[NodeLinks[i*LinksPerNode + j]].x, NodePosition[NodeLinks[i*LinksPerNode + j]].y, NodePosition[NodeLinks[i*LinksPerNode + j]].z);
				glEnd();
			}
			
		}	
	}
	
	glutSwapBuffers();
}

void generalMuscleForces()
{
	float f; 
	float dx, dy, dz, d;
	int muscleNumber, nodeNumber;
	
	// Getting forces on the nodes from the muscle fiber without contraction	
	for(int i = 0; i < NumberOfNodes; i++)
	{
		for(int j = 0; j < LinksPerNode; j++)
		{
			muscleNumber = NodeMuscles[i*LinksPerNode + j];
			nodeNumber = NodeLinks[i*LinksPerNode + j];
			if(nodeNumber != -1)
			{
				dx = NodePosition[nodeNumber].x - NodePosition[i].x;
				dy = NodePosition[nodeNumber].y - NodePosition[i].y;
				dz = NodePosition[nodeNumber].z - NodePosition[i].z;
				d  = sqrt(dx*dx + dy*dy + dz*dz);
				
				// Grabbing numeric overflow before it happens.
				if(d < 0.00001) 
				{
					printf("\n TSU Error: In generalMuscleForces d is very small between nodeNumbers = %d %d seperation = %f. Take a look at this!\n", i, nodeNumber, d);
					glColor3d(0.0,0.0,1.0);
					
					// Displaying where the problem occured.
					glPushMatrix();
					glTranslatef(NodePosition[i].x, NodePosition[i].y, NodePosition[i].z);
					glutSolidSphere(0.03,20,20);
					glPopMatrix();
					glPushMatrix();
					glTranslatef(NodePosition[nodeNumber].x, NodePosition[nodeNumber].y, NodePosition[nodeNumber].z);
					glutSolidSphere(0.03,20,20);
					glPopMatrix();
					glutSwapBuffers();
					while(1);
				}
				if(d < MuscleCompresionStopFraction[muscleNumber]*MuscleLength[muscleNumber])
				{
					f  = MuscleRelaxedStrength[muscleNumber]*MuscleCompresionMultiplier*(d - MuscleLength[muscleNumber]);
				}
				else if(d < MuscleLength[muscleNumber])
				{
					f  = MuscleRelaxedStrength[muscleNumber]*(d - MuscleLength[muscleNumber]);
				}
				else
				{
					f  = MuscleRelaxedStrength[muscleNumber]*MuscleTentionMultiplier*(d - MuscleLength[muscleNumber]);
				}
				NodeForce[i].x  += f*dx/d;
				NodeForce[i].y  += f*dy/d;
				NodeForce[i].z  += f*dz/d;
			}
		}
	}
}

void outwardPresure()
{
	float f; 
	float dx, dy, dz, d;
	double centerOfMassX, centerOfMassY, centerOfMassZ, mass;
	
	centerOfMassX = 0.0;
	centerOfMassY = 0.0;
	centerOfMassZ = 0.0;
	mass = 0.0;
	for(int i = 0; i < NumberOfNodes; i++)
	{
		 centerOfMassX += NodePosition[i].x*NodeMass[i];
		 centerOfMassY += NodePosition[i].y*NodeMass[i];
		 centerOfMassZ += NodePosition[i].z*NodeMass[i];
		 mass += NodeMass[i];
	}
	centerOfMassX /= mass;
	centerOfMassY /= mass;
	centerOfMassZ /= mass;
	
	// Getting forces on the nodes from the presure of the blood pushing out	
	for(int i = 0; i < NumberOfNodes; i++)
	{
		dx = centerOfMassX - NodePosition[i].x;
		dy = centerOfMassY - NodePosition[i].y;
		dz = centerOfMassZ - NodePosition[i].z;
		d  = sqrt(dx*dx + dy*dy + dz*dz);
		
		// Grabbing numeric overflow before it happens.
		if(d < 0.0001) 
		{
			printf("\nTSU Error: In outwardPresure d is very small. Take a look at this\n");
			exit(0);
		}
		
		f  = -BloodPresure;
		
		NodeForce[i].x  += f*dx/d;
		NodeForce[i].y  += f*dy/d;
		NodeForce[i].z  += f*dz/d;
	}
}

void turnOnNodeMuscles(int index)
{
	for(int j = 0; j < LinksPerNode; j++)
	{
		if((NodeLinks[index*LinksPerNode + j] != -1) && (ContractionOnOff[NodeMuscles[index*LinksPerNode + j]] == 0))
		{
			ContractionOnOff[NodeMuscles[index*LinksPerNode + j]] = 1;
			ContractionTimer[NodeMuscles[index*LinksPerNode + j]] = 0.0;
		}
	}
}

int contractionForces(float dt, float time)
{
	float dx, dy, dz, d;
	int muscleNumber, nodeNumber;
	
	// Getting forces for the muscle fiber contraction
	for(int i = 0; i < NumberOfNodes; i++)
	{
		for(int j = 0; j < LinksPerNode; j++)
		{
			muscleNumber = NodeMuscles[i*LinksPerNode + j];
			nodeNumber = NodeLinks[i*LinksPerNode + j];
			if(nodeNumber != -1)
			{	
				if(ContractionOnOff[muscleNumber] == 1)
				{
					if((ActionPotentialDuration[muscleNumber] - dt < ContractionTimer[muscleNumber]) && (ContractionTimer[muscleNumber] < ActionPotentialDuration[muscleNumber] + dt))
					{
						if(NodeAblatedYesNo[nodeNumber] == 0)
						{
							turnOnNodeMuscles(nodeNumber);
						}
					}
					
					if(ContractionTimer[muscleNumber] < ActionPotentialDuration[muscleNumber])
					{
						MuscleColor[muscleNumber].x = 1.0;
						MuscleColor[muscleNumber].y = 1.0;
						MuscleColor[muscleNumber].z = 1.0;
					}
					else
					{
						MuscleColor[muscleNumber].x = 1.0;
						MuscleColor[muscleNumber].y = 0.0;
						MuscleColor[muscleNumber].z = 0.0;
					}
					
					if(ContractionTimer[muscleNumber] < ContractionDuration[muscleNumber])
					{
						dx = NodePosition[nodeNumber].x - NodePosition[i].x;
						dy = NodePosition[nodeNumber].y - NodePosition[i].y;
						dz = NodePosition[nodeNumber].z - NodePosition[i].z;
						d  = sqrt(dx*dx + dy*dy + dz*dz);
						
						// Grabbing numeric overflow before it happens.
						if(d < 0.00001) 
						{
							printf("\n TSU Error: In contractionForces d is very small between nodeNumbers = %d %d seperation = %f. Take a look at this\n", i, nodeNumber, d);
							glColor3d(0.0,0.0,1.0);
							
							// Displaying where the problem occured.
							glPushMatrix();
							glTranslatef(NodePosition[i].x, NodePosition[i].y, NodePosition[i].z);
							glutSolidSphere(0.03,20,20);
							glPopMatrix();
							glPushMatrix();
							glTranslatef(NodePosition[nodeNumber].x, NodePosition[nodeNumber].y, NodePosition[nodeNumber].z);
							glutSolidSphere(0.03,20,20);
							glPopMatrix();
							glutSwapBuffers();
							while(1);
						}
						
						NodeForce[i].x   += ContractionStrength[muscleNumber]*dx/d;
						NodeForce[i].y   += ContractionStrength[muscleNumber]*dy/d;
						NodeForce[i].z   += ContractionStrength[muscleNumber]*dz/d;
					
						ContractionTimer[muscleNumber] += dt;
					}
					else if(ContractionTimer[muscleNumber] < ContractionDuration[muscleNumber] + RelaxationDuration[muscleNumber])
					{
						NodeForce[i].x   += 0.0;
						NodeForce[i].y   += 0.0;
						NodeForce[i].z   += 0.0;
						
						ContractionTimer[muscleNumber] += dt;
					}
					else
					{
						ContractionOnOff[muscleNumber] = 0;
						ContractionTimer[muscleNumber] = 0.0;
					}
				}
			}
		}
	}
	return(1);
}

void dampingForce()
{
	for(int i = 0; i < NumberOfNodes; i++)
	{	
		NodeForce[i].x   += -Viscosity*NodeVelocity[i].x;
		NodeForce[i].y   += -Viscosity*NodeVelocity[i].y;
		NodeForce[i].z   += -Viscosity*NodeVelocity[i].z;
	}
}

void moveNodes(float dt, float time)  // LeapFrog
{
	// Moving the system forward in time with leap-frog.
	for(int i = 0; i < NumberOfNodes; i++)
	{
		if(time == 0.0)
		{
			NodeVelocity[i].x += (NodeForce[i].x/MuscleMass[i])*0.5*dt;
			NodeVelocity[i].y += (NodeForce[i].y/MuscleMass[i])*0.5*dt;
			NodeVelocity[i].z += (NodeForce[i].z/MuscleMass[i])*0.5*dt;
		}
		else
		{
			NodeVelocity[i].x += (NodeForce[i].x/MuscleMass[i])*dt;
			NodeVelocity[i].y += (NodeForce[i].y/MuscleMass[i])*dt;
			NodeVelocity[i].z += (NodeForce[i].z/MuscleMass[i])*dt;
		}

		NodePosition[i].x += NodeVelocity[i].x*dt;
		NodePosition[i].y += NodeVelocity[i].y*dt;
		NodePosition[i].z += NodeVelocity[i].z*dt;
	}
}

void n_body(float dt)
{	
	if(Pause != 1)
	{
		if(BeatPeriod <= BeatTimer)
		{
			turnOnNodeMuscles(0);
			BeatTimer = 0.0;
		}
		else BeatTimer += dt;
		
		hardCodedEctopicEvents(RunTime, dt);
		
		// Zeroing out the nodal forces.
		for(int i = 0; i < NumberOfNodes; i++)
		{
			NodeForce[i].x   = 0.0;
			NodeForce[i].y   = 0.0;
			NodeForce[i].z   = 0.0;
		}
		
		generalMuscleForces();
		
		contractionForces(dt, RunTime);
		
		outwardPresure();
		
		dampingForce();
		
		moveNodes(dt, RunTime);

		if(DrawTimer == DrawRate) 
		{
			drawPicture();
			DrawTimer = 0;
			printf("\n Time = %f", RunTime);
		}
		else DrawTimer++;
		
		RunTime += dt;
	}
}

void setup()
{	
	int type, divisions;
	
	printf("\n Entire the type of simulation you would like to run:");
	printf("\n 1 for a circle.");
	printf("\n 2 for a sphere.\n");
	scanf("%d",&type);
	
	if(type == 1)
	{
		printf("\n Entire the number of divisions you would like in your circle.\n");
		if(scanf("%d",&divisions) != 1)
		{
			printf("\n Input must be an integer. \n");
			printf("\n Good Bye. \n");
			exit(0);
		}
		
		if(divisions == 0)
		{
			printf("\n So you want to run a simulation with nothing in it.");
			printf("\n That's easy just look at a blank screen. \n");
			printf("\n Good Bye. \n");
			exit(0);
		}
		if(divisions == 1)
		{
			printf("\n Seriously a circle of 1!");
			printf("\n This is sad. You need to get out make some friends. \n");
			printf("\n Good Bye. \n");
			exit(0);
		}
	}
	else if(type == 2)
	{
		printf("\n Entire the number of divisions you would like in a great circle of your sphere.");
		printf("\n The number needs to be even and large than 4.\n");
		if(scanf("%d",&divisions) != 1)
		{
			printf("\n Input must be an integer. \n");
			printf("\n Good Bye. \n");
			exit(0);
		}
		if(divisions%2 != 0)
		{
			printf("\n I said the number had to be even!");
			printf("\n Beem me up Scotty. There is no intelligent life down here. \n");
			printf("\n Good Bye. \n");
			exit(0);
		}
		else if(divisions < 5)
		{
			printf("\n Yo Einstien! I said the number had to be greater than 4.\n");
			printf("\n Good Bye. \n");
			exit(0);
		}
	}
	else
	{
		printf("\n Type of simulation is incorrect. \n");
		printf("\n Good Bye. \n");
		exit(0);
	}
	printf("\n\n The Particle Modeling Group hopes you enjoy your simulation.r\n\n");
	printf("\n The simulation is paused. Move to the mouse over the simulation window and type the following commands.\n");
	printf("\n To run the simulation type r.");
	printf("\n To pause the simulation type p.");
	printf("\n The positive x-axis is to the right.");
	printf("\n The positive y-axis is up.");
	printf("\n The positive z-axis is towards you.");
	printf("\n For an orthoganal view type 0.");
	printf("\n For a fulsrum view type f");
	printf("\n To do a positive spin on the x-axis type x (negative X).");
	printf("\n To do a positive spin on the y-axis type y (negative Y).");
	printf("\n To do a positive spin on the Z-axis type z (negative Z).");
	printf("\n To zoom in type w (out W). Note zoom is meaningless in orthoganal mode.");
	printf("\n To center type c");
	printf("\n To center and out the sinus node up type n");
	printf("\n To ablate or unablate right click the mouse on the node you are interested in");
	printf("\n For best ablation results, pause the simulation and put it in orthaganal mode.");
	printf("\n To quit the simulation type q or hit the kill button on the window.");
	printf("\n\n Happy ablating!\n");
	
	
	allocateMemory(type, divisions);
	
	if(type == 1) setNodesAndMusclesCircle(divisions);
	else if(type == 2) setNodesAndMusclesSphere(divisions);
	
	linkNodesToMuscles();
	
	setMuscleAttributesAndNodeMasses(divisions);
	
	hardCodedAblatedNodes();
	
	DrawRate = 1000;
	BeatPeriod = 50;
	DrawTimer = 0; 
	RunTime = 0.0;
	BeatTimer = 0.0;
	Pause = 1;
}

int main(int argc, char** argv)
{
	XWindowSize = 1000;
	YWindowSize = 1000; 

	// Clip plains
	Near = 0.2;
	Far = 80.0;

	//Direction here your eye is located location
	EyeX = 0.0;
	EyeY = 0.0;
	EyeZ = 2.0;

	//Where you are looking
	CenterX = 0.0;
	CenterY = 0.0;
	CenterZ = 0.0;

	//Up vector for viewing
	UpX = 0.0;
	UpY = 1.0;
	UpZ = 0.0;
	
	setup();
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(5,5);
	Window = glutCreateWindow("SVT");
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
	glMatrixMode(GL_MODELVIEW);
	glClearColor(0.0, 0.0, 0.0, 0.0);
	
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutMouseFunc(mymouse);
	glutKeyboardFunc(KeyPressed);
	glutIdleFunc(idle);
	//setup();
	glutMainLoop();
	return 0;
}

