#include "hip/hip_runtime.h"
// nvcc SVT4.0.cu -o svt4.0 -lglut -lm -lGLU -lGL
//To stop hit "control c" in the window you launched it from. stuff

// Length will be in millimeters
// Time will be in milliseconds
// Mass will be in ???

// Fiber length 100 micrometers or 0.1 millimeters
// Sodium wave speed .5 meters/sec or 0.5 millimeters/millisec

#include <iostream>
#include <fstream>
#include <sstream>
#include <string.h>
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
using namespace std;

#define PI 3.141592654

//#define NumberOfNodes 266 //266 //62
//#define NumberOfMuscles 552 //552 //132
//#define LinksPerNode 24 //24

// Globals
float Dt;
int DrawRate;
int Pause;

int TypeOfShape;
int Divisions;

float Viscosity;
float BloodPresure;

float BeatPeriod;

float MassOfAtria;
float RadiusOfAtria = 1.0; // Should be 31.83098862

int NumberOfNodes;
int NumberOfMuscles;
int LinksPerNode;

float4 *NodePosition, *NodeVelocity, *NodeForce;
float *NodeMass;
int *NodeLinks; // The nodes that this node is connected to
int *NodeMuscles; // The muscle that connects this node to ther other nodes
float *NodeArea; // The surface area a node covers
int *NodeAblatedYesNo;

// How the muscle will act without contraction.
int *MuscleConectionA, *MuscleConectionB;
float *MuscleMass, *MuscleLength;
float *MuscleRelaxedStrength, BaseMuscleRelaxedStrength;
float *MuscleCompresionStopFraction, BaseMuscleCompresionStopFraction;
float3 *MuscleColor;
float MuscleCompresionMultiplier;
float MuscleTentionMultiplier;

// Muscle contraction parameters
int *ContractionOnOff;
float *ContractionTimer;
float *ActionPotentialSpeed, BaseActionPotentialSpeed; //0.5 is a good value.
float *ActionPotentialDuration;
float *ContractionDuration, BaseContractionDuration; // 100.0 is a good value
float *RelaxationDuration, BaseRelaxationDuration; // 200.0 is a good value
float *ContractionStrength, BaseContractionStrength; // 5.0 is a good value	

int   DrawTimer; 
float RunTime;
float BeatTimer;
float4 CenterOfSimulation;

// Window globals
static int Window;
int XWindowSize;
int YWindowSize; 
double Near;
double Far;
double EyeX;
double EyeY;
double EyeZ;
double CenterX;
double CenterY;
double CenterZ;
double UpX;
double UpY;
double UpZ;

// Prototyping functions
void allocateMemory(int, int);
void setNodesAndMusclesCircle(int); 
void setNodesAndMusclesSphere(int);
void linkNodesToMuscles();
void setMuscleAttributesAndNodeMasses(int, int);
void drawPicture();
void generalMuscleForces();
void outwardPresure();
void turnOnNodeMuscles(int);
int contractionForces(float, float);
void dampingForce();
void moveNodes(float, float);
void hardCodedAblatedNodes();
void hardCodedEctopicEvents(float, float);
void n_body(float);
void setup();
void KeyPressed(unsigned char, int, int);
void mymouse(int, int, int, int);
void Display(void);
void reshape(int, int);
void allocateMemory(int, int);
void simulationScript();
void readSimulationParameters();

#include "./setNodesAndMuscles.h"
#include "./callBackFunctions.h"

void readSimulationParameters()
{
/*
	MassOfAtria = 1.0;// Need to look this up. ????????????
	MuscleCompresionMultiplier = 50.0;  // How hard a muscle resists being compressed past its compression limit.
	MuscleTentionMultiplier = 50.0;  // How hard a musle pulls back when it is stretched past its natural length.
	Viscosity = 10.0; // Jsut something to give resistance to movement.This will be divided by the number of nodes for scalling. ????????
	BloodPresure = 1.0; This will be scaled by the number of noddes. ?????????????????
	
	BaseMuscleRelaxedStrength = 0.1; // This will be scaled by multiplying by muscle length. This is the standard but can be adjusted for each muscle.
	BaseMuscleCompresionStopFraction = 0.7; // The percentage a muscles length can contract. This is the standard but can be adjusted for each muscle.
	BaseActionPotentialSpeed = 0.2; // This is the speed of the action potential. This is the standard but can be adjusted for each muscle.	
		
	BaseContractionDuration = 20.0;  // 100.0
	BaseRelaxationDuration = 40.0;  // 200.0
	BaseContractionStrength = 0.2; // This will be scaled by multipling by muscle length.
*/
	
	ifstream data;
	string name;
	
	data.open("./simulationSetup");
	
	if(data.is_open() == 1)
	{
		getline(data,name,'=');
		data >> TypeOfShape;
		
		getline(data,name,'=');
		data >> Divisions;
		
		getline(data,name,'=');
		data >> Viscosity;
		
		getline(data,name,'=');
		data >> BloodPresure;
		
		getline(data,name,'=');
		data >> MassOfAtria;
		
		getline(data,name,'=');
		data >> MuscleCompresionMultiplier;
		
		getline(data,name,'=');
		data >> MuscleTentionMultiplier;
		
		getline(data,name,'=');
		data >> BaseMuscleRelaxedStrength;
		
		getline(data,name,'=');
		data >> BaseContractionStrength;
		
		getline(data,name,'=');
		data >> BaseMuscleCompresionStopFraction;
		
		getline(data,name,'=');
		data >> BaseContractionDuration;
		
		getline(data,name,'=');
		data >> BaseRelaxationDuration;
		
		getline(data,name,'=');
		data >> BaseActionPotentialSpeed;
		
		getline(data,name,'=');
		data >> BeatPeriod;
		
		getline(data,name,'=');
		data >> DrawRate;
		
		getline(data,name,'=');
		data >> Dt;
	}
	else
	{
		printf("\nTSU Error could not open simulationSetup file\n");
		exit(0);
	}
	data.close();
	
	if(TypeOfShape == 1)
	{	
		if(Divisions == 0)
		{
			printf("\n So you want to run a simulation with nothing in it.");
			printf("\n That's easy just look at a blank screen. \n");
			printf("\n Good Bye. \n");
			exit(0);
		}
		if(Divisions == 1)
		{
			printf("\n Seriously a circle of 1!");
			printf("\n This is sad. You need to get out make some friends. \n");
			printf("\n Good Bye. \n");
			exit(0);
		}
		printf("\n You will be simulating a circle with %d divisions\n", Divisions);
	}
	else if(TypeOfShape == 2)
	{
		if(Divisions%2 != 0)
		{
			printf("\n I said the number had to be even!");
			printf("\n Beem me up Scotty. There is no intelligent life down here. \n");
			printf("\n Good Bye. \n");
			exit(0);
		}
		else if(Divisions < 5)
		{
			printf("\n Yo Einstien! I said the number had to be even and greater than or equal to 4.\n");
			printf("\n Good Bye. \n");
			exit(0);
		}
	}
	else
	{
		printf("\n Type of simulation is incorrect. \n");
		printf("\n Good Bye. \n");
		exit(0);
	}
}

void allocateMemory(int type, int divisions)
{
	if(type == 1) // Circle
	{
		NumberOfNodes = divisions;
		NumberOfMuscles = divisions;
		LinksPerNode = 2;
	}
	else if(type == 2) //Sphere
	{
		NumberOfNodes = divisions*(divisions/2 - 1) + 2;
		NumberOfMuscles = divisions + (divisions*2)*(divisions/2 - 1);
		LinksPerNode = divisions;
	}
	else if(type == 3) //Sphere with thickness
	{
		printf("\n Thick spheres are not in yet.\n");
		exit(0);
	}
	else
	{
		printf("\n Bad object type.\n");
		exit(0);
	}
	
	NodePosition = (float4*)malloc(NumberOfNodes*sizeof(float4));
	NodeVelocity = (float4*)malloc(NumberOfNodes*sizeof(float4));
	NodeForce    = (float4*)malloc(NumberOfNodes*sizeof(float4));
	
	NodeMass = (float*)malloc(NumberOfNodes*sizeof(float));
	NodeArea = (float*)malloc(NumberOfNodes*sizeof(float));
	NodeLinks = (int*)malloc(NumberOfNodes*LinksPerNode*sizeof(int));
	NodeMuscles = (int*)malloc(NumberOfNodes*LinksPerNode*sizeof(int));
	NodeAblatedYesNo = (int*)malloc(NumberOfNodes*sizeof(int));

	// How the muscle will act without contraction.
	MuscleConectionA = (int*)malloc(NumberOfMuscles*sizeof(int));
	MuscleConectionB = (int*)malloc(NumberOfMuscles*sizeof(int));
	MuscleMass = (float*)malloc(NumberOfMuscles*sizeof(float));
	MuscleLength = (float*)malloc(NumberOfMuscles*sizeof(float));
	MuscleRelaxedStrength = (float*)malloc(NumberOfMuscles*sizeof(float));
	MuscleCompresionStopFraction = (float*)malloc(NumberOfMuscles*sizeof(float));
	MuscleColor = (float3*)malloc(NumberOfMuscles*sizeof(float3));


	// Muscle contraction parameters
	ContractionOnOff = (int*)malloc(NumberOfMuscles*sizeof(int));
	ContractionTimer = (float*)malloc(NumberOfMuscles*sizeof(float));
	ActionPotentialSpeed = (float*)malloc(NumberOfMuscles*sizeof(float));
	ActionPotentialDuration = (float*)malloc(NumberOfMuscles*sizeof(float));
	ContractionDuration = (float*)malloc(NumberOfMuscles*sizeof(float));
	RelaxationDuration = (float*)malloc(NumberOfMuscles*sizeof(float));
	ContractionStrength = (float*)malloc(NumberOfMuscles*sizeof(float));
}

void linkNodesToMuscles()
{
	//Setting the ends of the muscles
	int index = 0;
	for(int i = 0; i < NumberOfNodes; i++)
	{
		for(int j = 0; j < LinksPerNode; j++)
		{
			if(NodeLinks[i*LinksPerNode + j] != -1)
			{
				if(i < NodeLinks[i*LinksPerNode + j])
				{
					if(NumberOfMuscles <= index)
					{
						printf("\nTSU Error: number of muscles is out of bounds\n");
						exit(0);
					} 
					MuscleConectionA[index] = i;
					MuscleConectionB[index] = NodeLinks[i*LinksPerNode + j];
					index++;
				}
			}
		}
	}
	
	// Setting the node muscles. Each node will have a list of nodes they are attached to (NodeLinks[][]) and the muscle that attaches it to that node (NodeMuscles[][]).
	// Setting them all to -1 first.
	for(int i = 0; i < NumberOfNodes; i++)
	{
		for(int j = 0; j < LinksPerNode; j++)
		{
			NodeMuscles[i*LinksPerNode + j] = -1;
		}	
	}
	for(int i = 0; i < NumberOfNodes; i++)
	{
		for(int j = 0; j < LinksPerNode; j++)
		{
			if(NodeLinks[i*LinksPerNode + j] != -1)
			{
				for(int k = 0; k < NumberOfMuscles; k++)
				{
					if((MuscleConectionA[k] == i && MuscleConectionB[k] == NodeLinks[i*LinksPerNode + j]) || (MuscleConectionA[k] == NodeLinks[i*LinksPerNode + j] && MuscleConectionB[k] == i))
					{
						if(NumberOfNodes*LinksPerNode <= (i*LinksPerNode + j))
						{
							printf("\nTSU Error: number of muscles is out of bounds\n");
							exit(0);
						} 
						NodeMuscles[i*LinksPerNode + j] = k;
					}
				}
			}
		}
	}
}

void setMuscleAttributesAndNodeMasses(int type, int divisions)
{	
	float dx, dy, dz, d, d1, d2;
	float sum, totalLengthOfAllMuscles;
	float bloodPresureScaling;
	float surfaceArea;
	
	Viscosity /= NumberOfNodes;
	
	/*
	for(int i = 0; i < NumberOfNodes; i++)
	{
		dx = NodePosition[NodeLinks[i*LinksPerNode + 0]].x - NodePosition[i*LinksPerNode + 3].x;
		dy = NodePosition[NodeLinks[i*LinksPerNode + 0]].y - NodePosition[i*LinksPerNode + 3].y;
		dz = NodePosition[NodeLinks[i*LinksPerNode + 0]].z - NodePosition[i*LinksPerNode + 3].z;
		d1 = sqrt(dx*dx + dy*dy + dz*dz)/2.0;
		dx = NodePosition[NodeLinks[i*LinksPerNode + 1]].x - NodePosition[i*LinksPerNode + 2].x;
		dy = NodePosition[NodeLinks[i*LinksPerNode + 1]].y - NodePosition[i*LinksPerNode + 2].y;
		dz = NodePosition[NodeLinks[i*LinksPerNode + 1]].z - NodePosition[i*LinksPerNode + 2].z;
		d2 = sqrt(dx*dx + dy*dy + dz*dz)/2.0;
		NodeArea[i] = d1*d2;
		printf("\n node area[%d] = %f", i, NodeArea[i]);
	}
	*/
	
	surfaceArea = 4.0*PI*RadiusOfAtria*RadiusOfAtria;
	bloodPresureScaling = surfaceArea/NumberOfNodes; // Need to scale by density too ??????????????
	BloodPresure *= bloodPresureScaling;
	
	CenterOfSimulation.x = 0.0;
	CenterOfSimulation.y = 0.0;
	CenterOfSimulation.z = 0.0;
	
	//Finding the length of each muscle and the total length of all muscles.
	totalLengthOfAllMuscles = 0.0;
	for(int i = 0; i < NumberOfMuscles; i++)
	{	
		dx = NodePosition[MuscleConectionA[i]].x - NodePosition[MuscleConectionB[i]].x;
		dy = NodePosition[MuscleConectionA[i]].y - NodePosition[MuscleConectionB[i]].y;
		dz = NodePosition[MuscleConectionA[i]].z - NodePosition[MuscleConectionB[i]].z;
		d = sqrt(dx*dx + dy*dy + dz*dz);
		MuscleLength[i] = d;
		totalLengthOfAllMuscles += d;
	}
	
	// Setting the mass of all muscles.
	if(type == 1)
	{
		MassOfAtria /= divisions; // If you are on a circle. There are division circle that make up the sphere so the circle is 1/divsiions the total mass.
	}
	for(int i = 0; i < NumberOfMuscles; i++)
	{	
		MuscleMass[i] = (MuscleLength[i]/totalLengthOfAllMuscles)*MassOfAtria;
	}
	
	// Setting other parameters
	for(int i = 0; i < NumberOfMuscles; i++)
	{	
		MuscleRelaxedStrength[i] = BaseMuscleRelaxedStrength*MuscleLength[i];
		MuscleCompresionStopFraction[i] = BaseMuscleCompresionStopFraction;
		ContractionOnOff[i] = 0;
		ContractionTimer[i] = 0.0;
		ActionPotentialSpeed[i] = BaseActionPotentialSpeed; //0.2; // 0.2
		ActionPotentialDuration[i] = MuscleLength[i]/ActionPotentialSpeed[i];
		ContractionDuration[i] = BaseContractionDuration; //20.0;  // 100.0
		RelaxationDuration[i] = BaseRelaxationDuration; //40.0;  // 200.0
		ContractionStrength[i] = BaseContractionStrength*MuscleLength[i]; //0.1;
		
		MuscleColor[i].x = 1.0;
		MuscleColor[i].y = 0.0;
		MuscleColor[i].z = 0.0;
	}
	
	for(int i = 0; i < NumberOfNodes; i++)
	{
		NodeAblatedYesNo[i] = 0; // Setting all nodes to not ablated.
	}
	
	// Setting the node masses
	for(int i = 0; i < NumberOfNodes; i++)
	{
		sum = 0.0;
		for(int j = 0; j < LinksPerNode; j++)
		{
			if(NodeMuscles[i*LinksPerNode + j] != -1)
			{
				sum += MuscleMass[NodeMuscles[i*LinksPerNode + j]];
			}
		}
		NodeMass[i] = sum/2.0;
	}
}

void hardCodedAblatedNodes()
{
	//Nodes to ablate
	for(int i = 1; i < 23; i++)
	{	
		//NodeAblatedYesNo[i] = 1;
	}
	
	//NodeAblatedYesNo[49] = 1;
}

void hardCodedEctopicEvents(float time, float dt)
{
	float er = dt/2.0;
	
	if((50.0 - er <= time) && (time < 220.0 + er))
	{
		//turnOnNodeMuscles(31);
	}
	
	if((51.0 - er <= time) && (time < 230.0 + er))
	{
		//turnOnNodeMuscles(41);
	}
	
	if((240.0 - er <= time) && (time < 230.0 + er))
	{
		//turnOnNodeMuscles(59);
	}
}

void drawPicture()
{
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	
	glColor3d(1.0,1.0,0.0);
	glPushMatrix();
	glTranslatef(NodePosition[0].x, NodePosition[0].y, NodePosition[0].z);
	glutSolidSphere(0.03,20,20);
	glPopMatrix();
	
	// Drawing nodes
	for(int i = 1; i < NumberOfNodes; i++)
	{
		if(NodeAblatedYesNo[i] == 0)
		{
			glColor3d(0.0,1.0,0.0);
		}
		else
		{
			glColor3d(1.0,1.0,1.0);
		}
		glPushMatrix();
		glTranslatef(NodePosition[i].x, NodePosition[i].y, NodePosition[i].z);
		glutSolidSphere(0.01,20,20);
		glPopMatrix();	
	}
	
	/*
	glColor3d(1.0,1.0,0.0);
	glPushMatrix();
	glTranslatef(NodePosition[NumberOfNodes-1].x, NodePosition[NumberOfNodes-1].y, NodePosition[NumberOfNodes-1].z);
	glutSolidSphere(0.03,20,20);
	glPopMatrix();
	*/
	
	// Drawing muscles
	glColor3d(1.0,0.0,0.0);
	for(int i = 0; i < NumberOfNodes; i++)
	{
		for(int j = 0; j < LinksPerNode; j++)
		{
			if(NodeLinks[i*LinksPerNode + j] != -1)
			{
				//glColor3d(MuscleColor[NodeMuscles[i*LinksPerNode + j]].x, MuscleColor[NodeMuscles[i*LinksPerNode + j]].y, MuscleColor[NodeMuscles[i*LinksPerNode + j]].z);
				//glLineWidth(1.0/(Px[i]-Px[NodeLinks[i*LinksPerNode + j]]));
				glBegin(GL_LINES);
					glVertex3f(NodePosition[i].x, NodePosition[i].y, NodePosition[i].z);
					glVertex3f(NodePosition[NodeLinks[i*LinksPerNode + j]].x, NodePosition[NodeLinks[i*LinksPerNode + j]].y, NodePosition[NodeLinks[i*LinksPerNode + j]].z);
				glEnd();
			}
			
		}	
	}
	glutSwapBuffers();
}

void generalMuscleForces()
{
	float f; 
	float dx, dy, dz, d;
	int muscleNumber, nodeNumber;
	
	// Getting forces on the nodes from the muscle fiber without contraction	
	for(int i = 0; i < NumberOfNodes; i++)
	{
		for(int j = 0; j < LinksPerNode; j++)
		{
			muscleNumber = NodeMuscles[i*LinksPerNode + j];
			nodeNumber = NodeLinks[i*LinksPerNode + j];
			if(nodeNumber != -1)
			{
				dx = NodePosition[nodeNumber].x - NodePosition[i].x;
				dy = NodePosition[nodeNumber].y - NodePosition[i].y;
				dz = NodePosition[nodeNumber].z - NodePosition[i].z;
				d  = sqrt(dx*dx + dy*dy + dz*dz);
				
				// Grabbing numeric overflow before it happens.
				if(d < 0.00001) 
				{
					printf("\n TSU Error: In generalMuscleForces d is very small between nodeNumbers %d and %d the seperation is %f. Take a look at this!\n", i, nodeNumber, d);
					glColor3d(0.0,0.0,1.0);
					
					// Displaying where the problem occured.
					glPushMatrix();
					glTranslatef(NodePosition[i].x, NodePosition[i].y, NodePosition[i].z);
					glutSolidSphere(0.03,20,20);
					glPopMatrix();
					glPushMatrix();
					glTranslatef(NodePosition[nodeNumber].x, NodePosition[nodeNumber].y, NodePosition[nodeNumber].z);
					glutSolidSphere(0.03,20,20);
					glPopMatrix();
					glutSwapBuffers();
					while(1);
				}
				if(d < MuscleCompresionStopFraction[muscleNumber]*MuscleLength[muscleNumber])
				{
					f  = MuscleRelaxedStrength[muscleNumber]*MuscleCompresionMultiplier*(d - MuscleLength[muscleNumber]);
				}
				else if(d < MuscleLength[muscleNumber])
				{
					f  = MuscleRelaxedStrength[muscleNumber]*(d - MuscleLength[muscleNumber]);
				}
				else
				{
					f  = MuscleRelaxedStrength[muscleNumber]*MuscleTentionMultiplier*(d - MuscleLength[muscleNumber]);
				}
				NodeForce[i].x  += f*dx/d;
				NodeForce[i].y  += f*dy/d;
				NodeForce[i].z  += f*dz/d;
			}
		}
	}
}

void outwardPresure()
{
	float f; 
	float dx, dy, dz, d;
	double centerOfMassX, centerOfMassY, centerOfMassZ, mass;
	
	centerOfMassX = 0.0;
	centerOfMassY = 0.0;
	centerOfMassZ = 0.0;
	mass = 0.0;
	for(int i = 0; i < NumberOfNodes; i++)
	{
		 centerOfMassX += NodePosition[i].x*NodeMass[i];
		 centerOfMassY += NodePosition[i].y*NodeMass[i];
		 centerOfMassZ += NodePosition[i].z*NodeMass[i];
		 mass += NodeMass[i];
	}
	centerOfMassX /= mass;
	centerOfMassY /= mass;
	centerOfMassZ /= mass;
	
	// Getting forces on the nodes from the presure of the blood pushing out	
	for(int i = 0; i < NumberOfNodes; i++)
	{
		dx = centerOfMassX - NodePosition[i].x;
		dy = centerOfMassY - NodePosition[i].y;
		dz = centerOfMassZ - NodePosition[i].z;
		d  = sqrt(dx*dx + dy*dy + dz*dz);
		
		// Grabbing numeric overflow before it happens.
		if(d < 0.0001) 
		{
			printf("\nTSU Error: In outwardPresure d is very small. Take a look at this\n");
			exit(0);
		}
		
		f  = -BloodPresure;   //*NodeArea[i];
		
		NodeForce[i].x  += f*dx/d;
		NodeForce[i].y  += f*dy/d;
		NodeForce[i].z  += f*dz/d;
	}
}

void turnOnNodeMuscles(int index)
{
	for(int j = 0; j < LinksPerNode; j++)
	{
		if((NodeLinks[index*LinksPerNode + j] != -1) && (ContractionOnOff[NodeMuscles[index*LinksPerNode + j]] == 0))
		{
			ContractionOnOff[NodeMuscles[index*LinksPerNode + j]] = 1;
			ContractionTimer[NodeMuscles[index*LinksPerNode + j]] = 0.0;
		}
	}
}

int contractionForces(float dt, float time)
{
	float dx, dy, dz, d;
	int muscleNumber, nodeNumber;
	
	// Getting forces for the muscle fiber contraction
	for(int i = 0; i < NumberOfNodes; i++)
	{
		for(int j = 0; j < LinksPerNode; j++)
		{
			muscleNumber = NodeMuscles[i*LinksPerNode + j];
			nodeNumber = NodeLinks[i*LinksPerNode + j];
			if(nodeNumber != -1)
			{	
				if(ContractionOnOff[muscleNumber] == 1)
				{
					if((ActionPotentialDuration[muscleNumber] - dt < ContractionTimer[muscleNumber]) && (ContractionTimer[muscleNumber] < ActionPotentialDuration[muscleNumber] + dt))
					{
						if(NodeAblatedYesNo[nodeNumber] == 0)
						{
							turnOnNodeMuscles(nodeNumber);
						}
					}
					
					if(ContractionTimer[muscleNumber] < ActionPotentialDuration[muscleNumber])
					{
						MuscleColor[muscleNumber].x = 1.0;
						MuscleColor[muscleNumber].y = 1.0;
						MuscleColor[muscleNumber].z = 1.0;
					}
					else
					{
						MuscleColor[muscleNumber].x = 1.0;
						MuscleColor[muscleNumber].y = 0.0;
						MuscleColor[muscleNumber].z = 0.0;
					}
					
					if(ContractionTimer[muscleNumber] < ContractionDuration[muscleNumber])
					{
						dx = NodePosition[nodeNumber].x - NodePosition[i].x;
						dy = NodePosition[nodeNumber].y - NodePosition[i].y;
						dz = NodePosition[nodeNumber].z - NodePosition[i].z;
						d  = sqrt(dx*dx + dy*dy + dz*dz);
						
						// Grabbing numeric overflow before it happens.
						if(d < 0.00001) 
						{
							printf("\n TSU Error: In contractionForces d is very small between nodeNumbers = %d %d seperation = %f. Take a look at this\n", i, nodeNumber, d);
							glColor3d(0.0,0.0,1.0);
							
							// Displaying where the problem occured.
							glPushMatrix();
							glTranslatef(NodePosition[i].x, NodePosition[i].y, NodePosition[i].z);
							glutSolidSphere(0.03,20,20);
							glPopMatrix();
							glPushMatrix();
							glTranslatef(NodePosition[nodeNumber].x, NodePosition[nodeNumber].y, NodePosition[nodeNumber].z);
							glutSolidSphere(0.03,20,20);
							glPopMatrix();
							glutSwapBuffers();
							while(1);
						}
						
						NodeForce[i].x   += ContractionStrength[muscleNumber]*dx/d;
						NodeForce[i].y   += ContractionStrength[muscleNumber]*dy/d;
						NodeForce[i].z   += ContractionStrength[muscleNumber]*dz/d;
					
						ContractionTimer[muscleNumber] += dt;
					}
					else if(ContractionTimer[muscleNumber] < ContractionDuration[muscleNumber] + RelaxationDuration[muscleNumber])
					{
						NodeForce[i].x   += 0.0;
						NodeForce[i].y   += 0.0;
						NodeForce[i].z   += 0.0;
						
						ContractionTimer[muscleNumber] += dt;
					}
					else
					{
						ContractionOnOff[muscleNumber] = 0;
						ContractionTimer[muscleNumber] = 0.0;
					}
				}
			}
		}
	}
	return(1);
}

void dampingForce()
{
	for(int i = 0; i < NumberOfNodes; i++)
	{	
		NodeForce[i].x   += -Viscosity*NodeVelocity[i].x;
		NodeForce[i].y   += -Viscosity*NodeVelocity[i].y;
		NodeForce[i].z   += -Viscosity*NodeVelocity[i].z;
	}
}

void moveNodes(float dt, float time)  // LeapFrog
{
	// Moving the system forward in time with leap-frog.
	for(int i = 0; i < NumberOfNodes; i++)
	{
		if(time == 0.0)
		{
			NodeVelocity[i].x += (NodeForce[i].x/MuscleMass[i])*0.5*dt;
			NodeVelocity[i].y += (NodeForce[i].y/MuscleMass[i])*0.5*dt;
			NodeVelocity[i].z += (NodeForce[i].z/MuscleMass[i])*0.5*dt;
		}
		else
		{
			NodeVelocity[i].x += (NodeForce[i].x/MuscleMass[i])*dt;
			NodeVelocity[i].y += (NodeForce[i].y/MuscleMass[i])*dt;
			NodeVelocity[i].z += (NodeForce[i].z/MuscleMass[i])*dt;
		}

		NodePosition[i].x += NodeVelocity[i].x*dt;
		NodePosition[i].y += NodeVelocity[i].y*dt;
		NodePosition[i].z += NodeVelocity[i].z*dt;
	}
}

void n_body(float dt)
{	
	if(Pause != 1)
	{
		if(BeatPeriod <= BeatTimer)
		{
			turnOnNodeMuscles(0);
			BeatTimer = 0.0;
		}
		else BeatTimer += dt;
		
		hardCodedEctopicEvents(RunTime, dt);
		
		// Zeroing out the nodal forces.
		for(int i = 0; i < NumberOfNodes; i++)
		{
			NodeForce[i].x   = 0.0;
			NodeForce[i].y   = 0.0;
			NodeForce[i].z   = 0.0;
		}
		
		generalMuscleForces();
		
		contractionForces(dt, RunTime);
		
		outwardPresure();
		
		dampingForce();
		
		moveNodes(dt, RunTime);

		if(DrawTimer == DrawRate) 
		{
			drawPicture();
			DrawTimer = 0;
			printf("\n Time = %f", RunTime);
		}
		else DrawTimer++;
		
		RunTime += dt;
	}
}

void simulationScript()
{
	printf("\n\n\n The Particle Modeling Group hopes you injoy your interactive right atriam simulation.\n\n");
	printf("\n The simulation is paused.");
	printf("\n Move to the mouse over the simulation window and type the following commands.\n");
	printf("\n To run the simulation type r.");
	printf("\n To pause the simulation type p.");
	printf("\n The positive x-axis is to the right.");
	printf("\n The positive y-axis is up.");
	printf("\n The positive z-axis is towards you.");
	printf("\n For an orthoganal view type 0.");
	printf("\n For a fulstrum view type f");
	printf("\n To do a positive spin on the x-axis type x (negative X).");
	printf("\n To do a positive spin on the y-axis type y (negative Y).");
	printf("\n To do a positive spin on the Z-axis type z (negative Z).");
	printf("\n To zoom in type w (out W). Note zoom is meaningless in orthoganal mode.");
	printf("\n To center type c");
	printf("\n To center and out the sinus node up type n");
	printf("\n To ablate or unablate right click the mouse on the node you are interested in");
	printf("\n For best ablation results, pause the simulation and put it in orthaganal mode.");
	printf("\n To quit the simulation type q or hit the kill button on the window.");
	printf("\n\n Happy ablating!\n");
}

void setup()
{	
	readSimulationParameters();
	
	allocateMemory(TypeOfShape, Divisions);
	
	if(TypeOfShape == 1) setNodesAndMusclesCircle(Divisions);
	else if(TypeOfShape == 2) setNodesAndMusclesSphere(Divisions);
	
	linkNodesToMuscles();
	
	setMuscleAttributesAndNodeMasses(TypeOfShape, Divisions);
	
	hardCodedAblatedNodes();
	
	DrawTimer = 0; 
	RunTime = 0.0;
	BeatTimer = 0.0;
	Pause = 1;
	
	simulationScript();
}

int main(int argc, char** argv)
{
	XWindowSize = 1000;
	YWindowSize = 1000; 

	// Clip plains
	Near = 0.2;
	Far = 80.0;

	//Direction here your eye is located location
	EyeX = 0.0;
	EyeY = 0.0;
	EyeZ = 2.0;

	//Where you are looking
	CenterX = 0.0;
	CenterY = 0.0;
	CenterZ = 0.0;

	//Up vector for viewing
	UpX = 0.0;
	UpY = 1.0;
	UpZ = 0.0;
	
	//setup();
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(5,5);
	Window = glutCreateWindow("SVT");
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);
	glMatrixMode(GL_MODELVIEW);
	glClearColor(0.0, 0.0, 0.0, 0.0);
	
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutMouseFunc(mymouse);
	glutKeyboardFunc(KeyPressed);
	glutIdleFunc(idle);
	setup();
	glutMainLoop();
	return 0;
}

