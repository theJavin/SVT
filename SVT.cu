#include "hip/hip_runtime.h"
// nvcc SVT.cu -o svt -lglut -lm -lGLU -lGL
//To stop hit "control c" in the window you launched it from. stuff

// Length will be in millimeters
// Time will be in milliseconds
// Mass will be in ???

// Fiber length 100 micrometers or 0.1 millimeters
// Sodium wave speed .5 meters/sec or 0.5 millimeters/millisec
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define PI 3.141592654

#define XWindowSize 1000
#define YWindowSize 1000 

#define STOP_TIME 60000.0
#define DT  0.001

#define NUMBER_OF_NODES 62
#define NUMBER_OF_MUSCLES 132
	
#define DRAW_RATE 1000

// Globals
float4 NodePosition[NUMBER_OF_NODES], NodeVelocity[NUMBER_OF_NODES], NodeForce[NUMBER_OF_NODES];
float NodeMass[NUMBER_OF_NODES];
int NodeLinks[NUMBER_OF_NODES][12]; // The nodes that this node is connected to
int NodeMuscles[NUMBER_OF_NODES][12]; // The muscle that connects this node to ther other nodes

// How the muscle will act without contraction.
int MuscleConectionA[NUMBER_OF_MUSCLES];
int MuscleConectionB[NUMBER_OF_MUSCLES];
float MuscleMass[NUMBER_OF_MUSCLES];
float MuscleLength[NUMBER_OF_MUSCLES];
float MuscleRelaxedStrength[NUMBER_OF_MUSCLES];
float MuscleCompresionMultiplier = 10.0;
float MuscleTentionMultiplier = 10.0;
float MuscleCompresionStopFraction[NUMBER_OF_MUSCLES];  // 0.6 is the standard value
float Viscosity = 10.0;

// Muscle contraction parameters
int ContractionOnOff[NUMBER_OF_MUSCLES];
float ContractionTimer[NUMBER_OF_MUSCLES];
float ActionPotentialSpeed[NUMBER_OF_MUSCLES]; //0.5 is a good value.
float ActionPotentialDuration[NUMBER_OF_MUSCLES];
float ContractionDuration[NUMBER_OF_MUSCLES]; // 100.0 is a good value
float RelaxationDuration[NUMBER_OF_MUSCLES]; // 200.0 is a good value
float ContractionStrength[NUMBER_OF_MUSCLES]; // 5.0 is a good value

float BloodPresure = 0.1;

float BeatPeriod = 400.0;

int set_initial_conditions()
{	
	int index;
	float dx, dy, dz;
	float sum;
	
	// Node position values for a sphere with 62 nodes//0.5 is a good value.
	NodePosition[0].x = 0.0;
	NodePosition[0].y = 1.0;
	NodePosition[0].z = 0.0;
	NodePosition[NUMBER_OF_NODES-1].x = 0.0;
	NodePosition[NUMBER_OF_NODES-1].y = -1.0;
	NodePosition[NUMBER_OF_NODES-1].z = 0.0;
	
	index = 1;
	for(int i = 1; i < 6; i++)
	{
		for(int j = 0; j < 12; j++)
		{
			if((NUMBER_OF_NODES-1) <= index)
			{
				printf("\nTSU Error: number of nodes is out of bounds\n");
				return(0);
			} 
			NodePosition[index].y = sin(PI/2.0 -i*PI/6.0);
			NodePosition[index].x = cos(PI/2.0 -i*PI/6.0)*cos(j*PI/6.0);
			NodePosition[index].z = cos(PI/2.0 -i*PI/6.0)*sin(j*PI/6.0);
			
			index++;
		}	
	}
	
	// Zeroing out velocity and acceleration
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		NodeVelocity[index].y = 0.0;
		NodeVelocity[index].x = 0.0;
		NodeVelocity[index].z = 0.0;
		
		NodeForce[index].y = 0.0;
		NodeForce[index].x = 0.0;
		NodeForce[index].z = 0.0;
	}
	

	// Below are the edges for the links connecting the sphere.
	// 0: 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12
	
	// 1:  0  12 2  13		13: 1  24 14 25		25: 13 36 26 37
	// 2:  0  1  3  14		14: 2  13 15 26		26: 14 25 27 38
	// 3:  0  2  4  15		15: 3  14 16 27		27: 15 26 27 39
	// 4:  0  3  5  16		16: 4  15 17 28		28: 16 27 27 40
	// 5:  0  4  6  17		17: 5  16 18 29		29: 17 28 27 41
	// 6:  0  5  7  18		18: 6  17 19 30		30: 18 29 27 42
	// 7:  0  6  8  19		19: 7  18 20 31		31: 19 30 27 43
	// 8:  0  7  9  20		20: 8  19 21 32		32: 20 31 27 44
	// 9:  0  8  10 21		21: 9  20 22 33		33: 21 32 27 45
	// 10: 0  9  11 22		22: 10 21 23 34		34: 22 33 27 46
	// 11: 0  10 12 23		23: 11 22 24 35		35: 23 34 27 47
	// 12: 0  11 1  24		24: 12 23 13 36		36: 24 35 25 48
	
	
	// 37: 25 48 38 49
	// 38: 26 37 39 50
	// 39: 27 38 40 51
	// 40: 28 39 41 52
	// 41: 29 40 42 53
	// 42: 30 41 43 54
	// 43: 31 41 44 55
	// 44: 32 43 45 56
	// 45: 33 44 46 57
	// 46: 34 45 47 58
	// 47: 35 46 48 59
	// 48: 36 47 37 60
	
	// 49: 37 60 50 61
	// 50: 38 49 51 61
	// 51: 39 50 52 61
	// 52: 40 51 53 61
	// 53: 41 52 54 61
	// 54: 42 53 55 61
	// 55: 43 54 56 61
	// 56: 44 55 57 61
	// 57: 45 56 58 61
	// 58: 46 57 59 61
	// 59: 47 58 60 61
	// 60: 48 59 49 61
	
	// 61: 49 50 51 52 53 54 55 56 57 58 59 60
	
	// Setting the nodes to -1 so you can tell the nodes that where not used.
	// The first and the last nodes had 12 links so I made them all have 12.
	// The rest only had 4 so you may want to revisit this.

	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		for(int j = 0; j < 12; j++)
		{
			NodeLinks[i][j] =  -1;
			NodeMuscles[i][j] = -1;
		}	
	}
	
	// Setting edges for the 0th node.
	for(int i = 0; i < 12; i++)
	{
		NodeLinks[0][i] =  i + 1;
	}
	
	// Setting the edges that are connected to the 0th node
	for(int j = 0; j < 4; j++)
	{
		for(int i = 1; i < 13; i++)
		{
			// Connect to node above
			if(j == 0)
			{
				NodeLinks[i][j] =  0;
			}
			
			// Connect to the node to the left
			if(j == 1)
			{
				NodeLinks[i][j] =  (i+10)%12 + 1;
			}
			
			// Connect to the node to the right
			if(j == 2)
			{
				NodeLinks[i][j] =  (i+12)%12 + 1;
			}
			
			// Connect to the node below
			if(j == 3)
			{
				NodeLinks[i][j] =  i + 12;
			}
		}
		
		// Setting the middle 3 sections
		for(int k = 0; k <= 3*12; k += 12)
		{
			for(int i = 13 + k; i < 25 + k; i++)
			{
				// Connect to node above
				if(j == 0)
				{
					NodeLinks[i][j] =  i - 12;
				}
				
				// Connect to the node to the left
				if(j == 1)
				{
					NodeLinks[i][j] =  (i+10)%12 + 13 + k;
				}
				
				// Connect to the node to the right
				if(j == 2)
				{
					NodeLinks[i][j] =  (i+12)%12 + 13 + k;
				}
				
				// Connect to the node below
				if(j == 3)
				{
					NodeLinks[i][j] =  i + 12;
				}
			}
		}
		
		// Setting the edges that are linked to the 61st node
		for(int i = 49; i < 61; i++)
		{
			// Connect to node above
			if(j == 0)
			{
				NodeLinks[i][j] =  i - 12;
			}
			
			// Connect to the node to the left
			if(j == 1)
			{
				NodeLinks[i][j] =  (i+10)%12 + 49;
			}
			
			// Connect to the node to the right
			if(j == 2)
			{
				NodeLinks[i][j] =  (i+12)%12 + 49;
			}
			
			// Connect to the node below
			if(j == 3)
			{
				NodeLinks[i][j] =  NUMBER_OF_NODES - 1;
			}
		}
		
		// Setting the 61st node.
		for(int i = 0; i < 12; i++)
		{
			NodeLinks[61][i] =  i + 49;
		}
	}
	
	//Setting the ends of the muscles
	index = 0;
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		for(int j = 0; j < 12; j++)
		{
			if(NodeLinks[i][j] != -1)
			{
				if(i < NodeLinks[i][j])
				{
					if(NUMBER_OF_MUSCLES <= index)
					{
						printf("\nTSU Error: number of edges is out of bounds\n");
						return(0);
					} 
					MuscleConectionA[index] = i;
					MuscleConectionB[index] = NodeLinks[i][j];
					index++;
				}
			}
		}
	}
	
	// Setting the node muscles. Each node will have a list of nodes they are attached to (NodeLinks[][]) and the muscle that attaches it to that node (NodeMuscles[][]).
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		for(int j = 0; j < 12; j++)
		{
			if(NodeLinks[i][j] != -1)
			{
				for(int k = 0; k < NUMBER_OF_MUSCLES; k++)
				{
					if((MuscleConectionA[k] == i && MuscleConectionB[k] == NodeLinks[i][j]) || (MuscleConectionA[k] == NodeLinks[i][j] && MuscleConectionB[k] == i))
					{
						NodeMuscles[i][j] = k;
					}
				}
			}
		}
	}
	
	// Setting other parameters
	for(int i = 0; i < NUMBER_OF_MUSCLES; i++)
	{	
		MuscleMass[i] = 1.0;
		dx = NodePosition[MuscleConectionA[i]].x - NodePosition[MuscleConectionB[i]].x;
		dy = NodePosition[MuscleConectionA[i]].y - NodePosition[MuscleConectionB[i]].y;
		dz = NodePosition[MuscleConectionA[i]].z - NodePosition[MuscleConectionB[i]].z;
		MuscleLength[i] = sqrt(dx*dx + dy*dy + dz*dz);;
		MuscleRelaxedStrength[i] = 0.1;
		MuscleCompresionStopFraction[i] = 0.6;
		ContractionOnOff[i] = 0;
		ContractionTimer[i] = 0.0;
		ActionPotentialSpeed[i] = 0.5;
		ActionPotentialDuration[i] = ActionPotentialSpeed[i]/MuscleLength[i];
		ContractionDuration[i] = 100.0;
		RelaxationDuration[i] = 200.0;
		ContractionStrength[i] = 0.1;
	}
	
	// Setting the node masses
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		sum = 0.0;
		for(int j = 0; j < 12; j++)
		{
			if(NodeMuscles[i][j] != -1)
			{
				sum += MuscleMass[NodeMuscles[i][j]];
			}
		}
		NodeMass[i] = sum/2.0;
		printf("\nNodeMass[%d] = %f", i, NodeMass[i]);
	}

	return(1);
}

void draw_picture()
{
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	
	glColor3d(0.0,1.0,0.0);
	glPushMatrix();
	glTranslatef(NodePosition[0].x, NodePosition[0].y, NodePosition[0].z);
	glutSolidSphere(0.03,20,20);
	glPopMatrix();
	
	// Drawing nodes
	for(int i = 1; i < NUMBER_OF_NODES; i++)
	{
		glColor3d(1.0,1.0,1.0);
		glPushMatrix();
		glTranslatef(NodePosition[i].x, NodePosition[i].y, NodePosition[i].z);
		glutSolidSphere(0.01,20,20);
		glPopMatrix();	
	}
	
	// Drawing muscles
	glColor3d(1.0,0.0,0.0);
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		for(int j = 0; j < 12; j++)
		{
			if(NodeLinks[i][j] != -1)
			{
				//glLineWidth(1.0/(Px[i]-Px[NodeLinks[i][j]]));
				glBegin(GL_LINES);
					glVertex3f(NodePosition[i].x, NodePosition[i].y, NodePosition[i].z);
					glVertex3f(NodePosition[NodeLinks[i][j]].x, NodePosition[NodeLinks[i][j]].y, NodePosition[NodeLinks[i][j]].z);
				glEnd();
			}
			
		}	
	}

/*	
	// Drawing muscles
	glColor3d(1.0,0.0,0.0);
	for(int i = 0; i < N-1; i++)
	{
		glLineWidth(1.0/(Px[i+1]-Px[i]));
		glBegin(GL_LINES);
			glVertex3f(Px[i], 0.0, 0.0);
			glVertex3f(Px[i+1], 0.0, 0.0);
		glEnd();
	}

	// Drawing sodium wave front
	glColor3d(1.0,1.0,0.0);
	glPushMatrix();
	glTranslatef(APWaveFront, 0.0, 0.0);
	glutSolidSphere(0.02,20,20);
	glPopMatrix();
	/*	
	glColor3d(1.0,1.0,0.0);
	glLineWidth(2.0);
	glBegin(GL_LINES);
		glVertex3f(APWaveFront, -0.5, 0.0);
		glVertex3f(APWaveFront, 0.5, 0.0);
	glEnd();
	*/
	glutSwapBuffers();
}

void generalMuscleForces()
{
	float f; 
	float dx, dy, dz, d;
	int muscleNumber, nodeNumber;
	
	// Getting forces on the nodes from the muscle fiber without contraction	
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		for(int j = 0; j < 12; j++)
		{
			muscleNumber = NodeMuscles[i][j];
			nodeNumber = NodeLinks[i][j];
			if(nodeNumber != -1)
			{
				dx = NodePosition[nodeNumber].x - NodePosition[i].x;
				dy = NodePosition[nodeNumber].y - NodePosition[i].y;
				dz = NodePosition[nodeNumber].z - NodePosition[i].z;
				d  = sqrt(dx*dx + dy*dy + dz*dz);
				if(d < MuscleCompresionStopFraction[muscleNumber]*MuscleLength[muscleNumber])
				{
					f  = MuscleRelaxedStrength[muscleNumber]*MuscleCompresionMultiplier*(d - MuscleLength[muscleNumber]);
				}
				else if(d < MuscleLength[muscleNumber])
				{
					f  = MuscleRelaxedStrength[muscleNumber]*(d - MuscleLength[muscleNumber]);
				}
				else
				{
					f  = MuscleRelaxedStrength[muscleNumber]*MuscleTentionMultiplier*(d - MuscleLength[muscleNumber]);
				}
				NodeForce[i].x  += f*dx/d;
				NodeForce[i].y  += f*dy/d;
				NodeForce[i].z  += f*dz/d;
			}
		}
	}
}

void outwardPresure()
{
	float f; 
	float dx, dy, dz, d;
	
	// Getting forces on the nodes from the presure of the blood pushing out	
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		dx = 0.0 - NodePosition[i].x;
		dy = 0.0 - NodePosition[i].y;
		dz = 0.0 - NodePosition[i].z;
		d  = sqrt(dx*dx + dy*dy + dz*dz);
		
		f  = -BloodPresure;
		
		NodeForce[i].x  += f*dx/d;
		NodeForce[i].y  += f*dy/d;
		NodeForce[i].z  += f*dz/d;
	}
}

void turnOnNodeMuscles(int index)
{
	for(int j = 0; j < 12; j++)
	{
		if((NodeLinks[index][j] != -1) && (ContractionOnOff[NodeMuscles[index][j]] == 0))
		{
			ContractionOnOff[NodeMuscles[index][j]] = 1;
			ContractionTimer[NodeMuscles[index][j]] = 0.0;
		}
	}
}

int contractionForces(float dt, float time)
{
	float dx, dy, dz, d;
	int muscleNumber, nodeNumber;
	
	// Getting forces for the muscle fiber contraction
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		for(int j = 0; j < 12; j++)
		{
			muscleNumber = NodeMuscles[i][j];
			nodeNumber = NodeLinks[i][j];
			if(nodeNumber != -1)
			{	
				if(ContractionOnOff[muscleNumber] == 1)
				{
					if((ActionPotentialDuration[muscleNumber] - dt/2.0 < ContractionTimer[muscleNumber]) && (ContractionTimer[muscleNumber] < ActionPotentialDuration[muscleNumber] + dt/2.0))
					{
						turnOnNodeMuscles(nodeNumber);
					}
					if(ContractionTimer[muscleNumber] < ContractionDuration[muscleNumber])
					{
						dx = NodePosition[nodeNumber].x - NodePosition[i].x;
						dy = NodePosition[nodeNumber].y - NodePosition[i].y;
						dz = NodePosition[nodeNumber].z - NodePosition[i].z;
						d  = sqrt(dx*dx + dy*dy + dz*dz);
						
						NodeForce[i].x   += ContractionStrength[muscleNumber]*dx/d;
						NodeForce[i].y   += ContractionStrength[muscleNumber]*dy/d;
						NodeForce[i].z   += ContractionStrength[muscleNumber]*dz/d;
					
						ContractionTimer[muscleNumber] += dt;
					}
					else if(ContractionTimer[muscleNumber] < ContractionDuration[muscleNumber] + RelaxationDuration[muscleNumber])
					{
						NodeForce[i].x   += 0.0;
						NodeForce[i].y   += 0.0;
						NodeForce[i].z   += 0.0;
						ContractionTimer[muscleNumber] += dt;
					}
					else
					{
						ContractionOnOff[muscleNumber] = 0;
						ContractionTimer[muscleNumber] = 0.0;
					}
				}
			}
		}
	}
	return(1);
}

void dampingForce()
{
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{	
		NodeForce[i].x   += -Viscosity*NodeVelocity[i].x;
		NodeForce[i].y   += -Viscosity*NodeVelocity[i].y;
		NodeForce[i].z   += -Viscosity*NodeVelocity[i].z;
	}
}

void moveNodes(float dt, float time)  // LeapFrog
{
	// Moving the system forward in time with leap-frog.
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		if(time == 0.0)
		{
			NodeVelocity[i].x += (NodeForce[i].x/MuscleMass[i])*0.5*dt;
			NodeVelocity[i].y += (NodeForce[i].y/MuscleMass[i])*0.5*dt;
			NodeVelocity[i].z += (NodeForce[i].z/MuscleMass[i])*0.5*dt;
		}
		else
		{
			NodeVelocity[i].x += (NodeForce[i].x/MuscleMass[i])*dt;
			NodeVelocity[i].y += (NodeForce[i].y/MuscleMass[i])*dt;
			NodeVelocity[i].z += (NodeForce[i].z/MuscleMass[i])*dt;
		}

		NodePosition[i].x += NodeVelocity[i].x*dt;
		NodePosition[i].y += NodeVelocity[i].y*dt;
		NodePosition[i].z += NodeVelocity[i].z*dt;
	}
}

int n_body()
{
	int   tdraw = 0; 
	double time = 0.0;
	float beatTimer = 0.0;
	
	while(time < STOP_TIME)
	{
		if(BeatPeriod <= beatTimer)
		{
			turnOnNodeMuscles(0);
			beatTimer = 0.0;
		}
		else beatTimer += DT;
		
		// Zeroing out the nodal forces.
		for(int i = 0; i < NUMBER_OF_NODES; i++)
		{
			NodeForce[i].x   = 0.0;
			NodeForce[i].y   = 0.0;
			NodeForce[i].z   = 0.0;
		}
		
		generalMuscleForces();
		
		contractionForces(DT, time);
		
		outwardPresure();
		
		dampingForce();
		
		moveNodes(DT, time);

		if(tdraw == DRAW_RATE) 
		{
			draw_picture();
			tdraw = 0;
			printf("\n Time = %f", time);
		}
		else tdraw++;
		
		time += DT;
	}
	return(1);
}

void control()
{	
	//int    tdraw = 0;
	//float  time = 0.0;

	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);

	set_initial_conditions();
	
	draw_picture();
	
	if(n_body() == 1) printf("\n N-body success \n");
	
	printf("\n DONE \n");
	while(1);
}

void Display(void)
{
	gluLookAt(0.0, 2.0, 2.0, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0);
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	glutSwapBuffers();
	glFlush();
	control();
}

void reshape(int w, int h)
{
	glViewport(0, 0, (GLsizei) w, (GLsizei) h);

	glMatrixMode(GL_PROJECTION);

	glLoadIdentity();

	glFrustum(-0.2, 0.2, -0.2, 0.2, 0.2, 80.0);

	glMatrixMode(GL_MODELVIEW);
}

int main(int argc, char** argv)
{
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(0,0);
	glutCreateWindow("1D Myocardium");
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutMainLoop();
	return 0;
}

