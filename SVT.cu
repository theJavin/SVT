#include "hip/hip_runtime.h"
// nvcc SVT.cu -o svt -lglut -lm -lGLU -lGL
//To stop hit "control c" in the window you launched it from.

// Length will be in millimeters
// Time will be in milliseconds
// Mass will be in... mass units?

// Fiber length 100 micrometers or 0.1 millimeters
// Sodium wave speed .5 meters/sec or 0.5 millimeters/millisec

#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define PI 3.141592654

#define STOP_TIME 60000.0
#define DT  0.001

#define NUMBER_OF_NODES 266 //266 //62
#define NUMBER_OF_MUSCLES 552 //552 //132
#define LINKS_PER_NODE 24 //24

// Globals
int DrawRate;

float4 NodePosition[NUMBER_OF_NODES], NodeVelocity[NUMBER_OF_NODES], NodeForce[NUMBER_OF_NODES];
float NodeMass[NUMBER_OF_NODES];
int NodeLinks[NUMBER_OF_NODES][LINKS_PER_NODE]; // The nodes that this node is connected to
int NodeMuscles[NUMBER_OF_NODES][LINKS_PER_NODE]; // The muscle that connects this node to ther other nodes
int NodeAblatedYesNo[NUMBER_OF_NODES];

// How the muscle will act without contraction.
int MuscleConectionA[NUMBER_OF_MUSCLES];
int MuscleConectionB[NUMBER_OF_MUSCLES];
float MuscleMass[NUMBER_OF_MUSCLES];
float MuscleLength[NUMBER_OF_MUSCLES];
float MuscleRelaxedStrength[NUMBER_OF_MUSCLES];
float MuscleCompresionMultiplier;
float MuscleTentionMultiplier;
float MuscleCompresionStopFraction[NUMBER_OF_MUSCLES];  // 0.6 is the standard value
float Viscosity;
float3 MuscleColor[NUMBER_OF_MUSCLES];

// Muscle contraction parameters
int ContractionOnOff[NUMBER_OF_MUSCLES];
float ContractionTimer[NUMBER_OF_MUSCLES];
float ActionPotentialSpeed[NUMBER_OF_MUSCLES]; //0.5 is a good value.
float ActionPotentialDuration[NUMBER_OF_MUSCLES];
float ContractionDuration[NUMBER_OF_MUSCLES]; // 100.0 is a good value
float RelaxationDuration[NUMBER_OF_MUSCLES]; // 200.0 is a good value
float ContractionStrength[NUMBER_OF_MUSCLES]; // 5.0 is a good value

float BloodPresure;
float BeatPeriod;

// Prototyping functions
void initializeNodesAndLinksSphere62();
void linkNodesToMuscles();
void setMuscleAtributesAndNodeMasses();
void ablatedNodes();
void draw_picture();
void generalMuscleForces();
void outwardPresure();
void turnOnNodeMuscles(int);
int contractionForces(float, float);
void dampingForce();
void moveNodes(float, float);
void ectopicEvents(float, float);
int n_body(float);
void control();
void mymouse(int, int, int, int);
void Display(void);
void reshape(int, int);

#include "./setNodesAndLinks.h"

void linkNodesToMuscles()
{
	//Setting the ends of the muscles
	int index = 0;
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		for(int j = 0; j < LINKS_PER_NODE; j++)
		{
			if(NodeLinks[i][j] != -1)
			{
				if(i < NodeLinks[i][j])
				{
					if(NUMBER_OF_MUSCLES <= index)
					{
						printf("\nTSU Error: number of muscles is out of bounds\n");
						exit(0);
					} 
					MuscleConectionA[index] = i;
					MuscleConectionB[index] = NodeLinks[i][j];
					index++;
				}
			}
		}
	}
	
	// Setting the node muscles. Each node will have a list of nodes they are attached to (NodeLinks[][]) and the muscle that attaches it to that node (NodeMuscles[][]).
	// Setting them all to -1 first.
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		for(int j = 0; j < LINKS_PER_NODE; j++)
		{
			NodeMuscles[i][j] = -1;
		}	
	}
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		for(int j = 0; j < LINKS_PER_NODE; j++)
		{
			if(NodeLinks[i][j] != -1)
			{
				for(int k = 0; k < NUMBER_OF_MUSCLES; k++)
				{
					if((MuscleConectionA[k] == i && MuscleConectionB[k] == NodeLinks[i][j]) || (MuscleConectionA[k] == NodeLinks[i][j] && MuscleConectionB[k] == i))
					{
						NodeMuscles[i][j] = k;
					}
				}
			}
		}
	}
}

void setMuscleAtributesAndNodeMasses()
{	
	float dx, dy, dz;
	float sum;
	
	MuscleCompresionMultiplier = 50.0;
	MuscleTentionMultiplier = 50.0;
	Viscosity = 5.0;
	BloodPresure = 0.02;
	
	// Setting other parameters
	for(int i = 0; i < NUMBER_OF_MUSCLES; i++)
	{	
		MuscleMass[i] = 1.0;
		dx = NodePosition[MuscleConectionA[i]].x - NodePosition[MuscleConectionB[i]].x;
		dy = NodePosition[MuscleConectionA[i]].y - NodePosition[MuscleConectionB[i]].y;
		dz = NodePosition[MuscleConectionA[i]].z - NodePosition[MuscleConectionB[i]].z;
		MuscleLength[i] = sqrt(dx*dx + dy*dy + dz*dz);;
		MuscleRelaxedStrength[i] = 0.1;
		MuscleCompresionStopFraction[i] = 0.6;
		ContractionOnOff[i] = 0;
		ContractionTimer[i] = 0.0;
		ActionPotentialSpeed[i] = 0.02; // 0.2
		ActionPotentialDuration[i] = MuscleLength[i]/ActionPotentialSpeed[i];
		ContractionDuration[i] = 20.0;  // 100.0
		RelaxationDuration[i] = 60.0;  // 200.0
		ContractionStrength[i] = 0.2; //0.1;
		MuscleColor[i].x = 1.0;
		MuscleColor[i].y = 0.0;
		MuscleColor[i].z = 0.0;
	}
	
	// Setting the node masses
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		sum = 0.0;
		for(int j = 0; j < LINKS_PER_NODE; j++)
		{
			if(NodeMuscles[i][j] != -1)
			{
				sum += MuscleMass[NodeMuscles[i][j]];
			}
		}
		NodeMass[i] = sum/2.0;
	}
}

void draw_picture()
{
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	
	glColor3d(1.0,1.0,0.0);
	glPushMatrix();
	glTranslatef(NodePosition[0].x, NodePosition[0].y, NodePosition[0].z);
	glutSolidSphere(0.03,20,20);
	glPopMatrix();
	
	// Drawing nodes
	for(int i = 1; i < NUMBER_OF_NODES; i++)
	{
		if(NodeAblatedYesNo[i] == 0)
		{
			glColor3d(0.0,1.0,0.0);
		}
		else
		{
			glColor3d(1.0,1.0,1.0);
		}
		glPushMatrix();
		glTranslatef(NodePosition[i].x, NodePosition[i].y, NodePosition[i].z);
		glutSolidSphere(0.01,20,20);
		glPopMatrix();	
	}
	
	/*
	glColor3d(1.0,1.0,0.0);
	glPushMatrix();
	glTranslatef(NodePosition[NUMBER_OF_NODES-1].x, NodePosition[NUMBER_OF_NODES-1].y, NodePosition[NUMBER_OF_NODES-1].z);
	glutSolidSphere(0.03,20,20);
	glPopMatrix();
	*/
	
	// Drawing muscles
	glColor3d(1.0,0.0,0.0);
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		for(int j = 0; j < LINKS_PER_NODE; j++)
		{
			if(NodeLinks[i][j] != -1)
			{
				glColor3d(MuscleColor[NodeMuscles[i][j]].x, MuscleColor[NodeMuscles[i][j]].y, MuscleColor[NodeMuscles[i][j]].z);
				//glLineWidth(1.0/(Px[i]-Px[NodeLinks[i][j]]));
				glBegin(GL_LINES);
					glVertex3f(NodePosition[i].x, NodePosition[i].y, NodePosition[i].z);
					glVertex3f(NodePosition[NodeLinks[i][j]].x, NodePosition[NodeLinks[i][j]].y, NodePosition[NodeLinks[i][j]].z);
				glEnd();
			}
			
		}	
	}
	
	glutSwapBuffers();
}

void generalMuscleForces()
{
	float f; 
	float dx, dy, dz, d;
	int muscleNumber, nodeNumber;
	
	// Getting forces on the nodes from the muscle fiber without contraction	
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		for(int j = 0; j < LINKS_PER_NODE; j++)
		{
			muscleNumber = NodeMuscles[i][j];
			nodeNumber = NodeLinks[i][j];
			if(nodeNumber != -1)
			{
				dx = NodePosition[nodeNumber].x - NodePosition[i].x;
				dy = NodePosition[nodeNumber].y - NodePosition[i].y;
				dz = NodePosition[nodeNumber].z - NodePosition[i].z;
				d  = sqrt(dx*dx + dy*dy + dz*dz);
				if(d < 0.00001) 
				{
					printf("\n TSU Error: In generalMuscleForces d is very small between nodeNumbers = %d %d seperation = %f. Take a look at this\n", i, nodeNumber, d);
					glColor3d(0.0,0.0,1.0);
					glPushMatrix();
					glTranslatef(NodePosition[i].x, NodePosition[i].y, NodePosition[i].z);
					glutSolidSphere(0.03,20,20);
					glPopMatrix();
					glPushMatrix();
					glTranslatef(NodePosition[nodeNumber].x, NodePosition[nodeNumber].y, NodePosition[nodeNumber].z);
					glutSolidSphere(0.03,20,20);
					glPopMatrix();
					glutSwapBuffers();
					while(1);
				}
				if(d < MuscleCompresionStopFraction[muscleNumber]*MuscleLength[muscleNumber])
				{
					f  = MuscleRelaxedStrength[muscleNumber]*MuscleCompresionMultiplier*(d - MuscleLength[muscleNumber]);
				}
				else if(d < MuscleLength[muscleNumber])
				{
					f  = MuscleRelaxedStrength[muscleNumber]*(d - MuscleLength[muscleNumber]);
				}
				else
				{
					f  = MuscleRelaxedStrength[muscleNumber]*MuscleTentionMultiplier*(d - MuscleLength[muscleNumber]);
				}
				NodeForce[i].x  += f*dx/d;
				NodeForce[i].y  += f*dy/d;
				NodeForce[i].z  += f*dz/d;
			}
		}
	}
}

void outwardPresure()
{
	float f; 
	float dx, dy, dz, d;
	float4 centerOfMass;
	
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		 centerOfMass.x += NodePosition[i].x*NodeMass[i];
		 centerOfMass.y += NodePosition[i].y*NodeMass[i];
		 centerOfMass.z += NodePosition[i].z*NodeMass[i];
		 centerOfMass.w += NodeMass[i];
	}
	
	centerOfMass.x /= centerOfMass.w;
	centerOfMass.y /= centerOfMass.w;
	centerOfMass.z /= centerOfMass.w;
		 
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		 NodePosition[i].x -= centerOfMass.x;
		 NodePosition[i].y -= centerOfMass.y;
		 NodePosition[i].z -= centerOfMass.z;
	}
	
	// Getting forces on the nodes from the presure of the blood pushing out	
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		dx = 0.0 - NodePosition[i].x;
		dy = 0.0 - NodePosition[i].y;
		dz = 0.0 - NodePosition[i].z;
		d  = sqrt(dx*dx + dy*dy + dz*dz);
		if(d < 0.0001) 
		{
			printf("\nTSU Error: In outwardPresure d is very small. Take a look at this\n");
			exit(0);
		}
		
		f  = -BloodPresure;
		
		NodeForce[i].x  += f*dx/d;
		NodeForce[i].y  += f*dy/d;
		NodeForce[i].z  += f*dz/d;
	}
}

void turnOnNodeMuscles(int index)
{
	for(int j = 0; j < LINKS_PER_NODE; j++)
	{
		if((NodeLinks[index][j] != -1) && (ContractionOnOff[NodeMuscles[index][j]] == 0))
		{
			ContractionOnOff[NodeMuscles[index][j]] = 1;
			ContractionTimer[NodeMuscles[index][j]] = 0.0;
		}
	}
}

int contractionForces(float dt, float time)
{
	float dx, dy, dz, d;
	int muscleNumber, nodeNumber;
	
	// Getting forces for the muscle fiber contraction
	for(int i = 0; i < NUMBER_OF_NODES; i++) 
	{
		for(int j = 0; j < LINKS_PER_NODE; j++)
		{
			muscleNumber = NodeMuscles[i][j];
			nodeNumber = NodeLinks[i][j];
			if(nodeNumber != -1)
			{	
				if(ContractionOnOff[muscleNumber] == 1)
				{
					if((ActionPotentialDuration[muscleNumber] - dt < ContractionTimer[muscleNumber]) && (ContractionTimer[muscleNumber] < ActionPotentialDuration[muscleNumber] + dt))
					{
						if(NodeAblatedYesNo[nodeNumber] == 0)
						{
							turnOnNodeMuscles(nodeNumber);
						}
					}
					
					if(ContractionTimer[muscleNumber] < ActionPotentialDuration[muscleNumber])
					{
						MuscleColor[muscleNumber].x = 1.0;
						MuscleColor[muscleNumber].y = 1.0;
						MuscleColor[muscleNumber].z = 1.0;
					}
					else
					{
						MuscleColor[muscleNumber].x = 1.0;
						MuscleColor[muscleNumber].y = 0.0;
						MuscleColor[muscleNumber].z = 0.0;
					}
					
					if(ContractionTimer[muscleNumber] < ContractionDuration[muscleNumber])
					{
						dx = NodePosition[nodeNumber].x - NodePosition[i].x;
						dy = NodePosition[nodeNumber].y - NodePosition[i].y;
						dz = NodePosition[nodeNumber].z - NodePosition[i].z;
						d  = sqrt(dx*dx + dy*dy + dz*dz);
						if(d < 0.00001) 
						{
							printf("\n TSU Error: In contractionForces d is very small between nodeNumbers = %d %d seperation = %f. Take a look at this\n", i, nodeNumber, d);
							glColor3d(0.0,0.0,1.0);
							glPushMatrix();
							glTranslatef(NodePosition[i].x, NodePosition[i].y, NodePosition[i].z);
							glutSolidSphere(0.03,20,20);
							glPopMatrix();
							glPushMatrix();
							glTranslatef(NodePosition[nodeNumber].x, NodePosition[nodeNumber].y, NodePosition[nodeNumber].z);
							glutSolidSphere(0.03,20,20);
							glPopMatrix();
							glutSwapBuffers();
							while(1);
						}
						
						NodeForce[i].x   += ContractionStrength[muscleNumber]*dx/d;
						NodeForce[i].y   += ContractionStrength[muscleNumber]*dy/d;
						NodeForce[i].z   += ContractionStrength[muscleNumber]*dz/d;
					
						ContractionTimer[muscleNumber] += dt;
					}
					else if(ContractionTimer[muscleNumber] < ContractionDuration[muscleNumber] + RelaxationDuration[muscleNumber])
					{
						NodeForce[i].x   += 0.0;
						NodeForce[i].y   += 0.0;
						NodeForce[i].z   += 0.0;
						
						ContractionTimer[muscleNumber] += dt;
					}
					else
					{
						ContractionOnOff[muscleNumber] = 0;
						ContractionTimer[muscleNumber] = 0.0;
					}
				}
			}
		}
	}
	return(1);
}

void dampingForce()
{
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{	
		NodeForce[i].x   += -Viscosity*NodeVelocity[i].x;
		NodeForce[i].y   += -Viscosity*NodeVelocity[i].y;
		NodeForce[i].z   += -Viscosity*NodeVelocity[i].z;
	}
}

void moveNodes(float dt, float time)  // LeapFrog
{
	// Moving the system forward in time with leap-frog.
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		if(time == 0.0)
		{
			NodeVelocity[i].x += (NodeForce[i].x/MuscleMass[i])*0.5*dt;
			NodeVelocity[i].y += (NodeForce[i].y/MuscleMass[i])*0.5*dt;
			NodeVelocity[i].z += (NodeForce[i].z/MuscleMass[i])*0.5*dt;
		}
		else
		{
			NodeVelocity[i].x += (NodeForce[i].x/MuscleMass[i])*dt;
			NodeVelocity[i].y += (NodeForce[i].y/MuscleMass[i])*dt;
			NodeVelocity[i].z += (NodeForce[i].z/MuscleMass[i])*dt;
		}

		NodePosition[i].x += NodeVelocity[i].x*dt;
		NodePosition[i].y += NodeVelocity[i].y*dt;
		NodePosition[i].z += NodeVelocity[i].z*dt;
	}
}

int n_body(float dt)
{
	int   tdraw = 0; 
	double time = 0.0;
	float beatTimer = 0.0;
	
	while(time < STOP_TIME)
	{
		if(BeatPeriod <= beatTimer)
		{
			turnOnNodeMuscles(0);
			beatTimer = 0.0;
		}
		else beatTimer += dt;
		
		ectopicEvents(time, dt);
		
		// Zeroing out the nodal forces.
		for(int i = 0; i < NUMBER_OF_NODES; i++)
		{
			NodeForce[i].x   = 0.0;
			NodeForce[i].y   = 0.0;
			NodeForce[i].z   = 0.0;
		}
		
		generalMuscleForces();
		
		contractionForces(dt, time);
		
		outwardPresure();
		
		dampingForce();
		
		moveNodes(dt, time);

		if(tdraw == DrawRate) 
		{
			draw_picture();
			tdraw = 0;
			printf("\n Time = %f", time);
		}
		else tdraw++;
		
		time += dt;
	}
	return(1);
}

void ablatedNodes()
{
	// Setting all nodes as not ablated
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		NodeAblatedYesNo[i] = 0;
	}
	
	//Nodes to ablate
	for(int i = 1; i < 23; i++)
	{	
		//NodeAblatedYesNo[i] = 1;
	}
	
	
	NodeAblatedYesNo[1] = 1;
	NodeAblatedYesNo[13] = 1;
	NodeAblatedYesNo[25] = 1;
	NodeAblatedYesNo[37] = 1;
	NodeAblatedYesNo[49] = 1;
	
	NodeAblatedYesNo[2] = 1;
	NodeAblatedYesNo[3] = 1;
	NodeAblatedYesNo[4] = 1;
	NodeAblatedYesNo[5] = 1;
	NodeAblatedYesNo[6] = 1;
	NodeAblatedYesNo[7] = 1;
	NodeAblatedYesNo[8] = 1;
	
	NodeAblatedYesNo[9] = 1;
	NodeAblatedYesNo[10] = 1;
	NodeAblatedYesNo[11] = 1;
	NodeAblatedYesNo[14] = 1;
	NodeAblatedYesNo[17] = 1;
	
}

void ectopicEvents(float time, float dt)
{
	float er = dt/2.0;
	
	if((50.0 - er <= time) && (time < 220.0 + er))
	{
		turnOnNodeMuscles(31);
	}
	
	if((51.0 - er <= time) && (time < 230.0 + er))
	{
		//turnOnNodeMuscles(41);
	}
	
	if((240.0 - er <= time) && (time < 230.0 + er))
	{
		//turnOnNodeMuscles(59);
	}
}

void control()
{	
	
	//initializeNodesAndLinksSphere62();
	//initializeNodesAndLinksSphere266();
	initializeNodesAndLinksSphere(24);
	
	linkNodesToMuscles();
	setMuscleAtributesAndNodeMasses();
	ablatedNodes();
	
	draw_picture();
	
	DrawRate = 1000;
	BeatPeriod = 100;
	
	if(n_body(DT) == 1) printf("\n N-body success \n");
	
	printf("\n DONE \n");
	while(1);
}

// Window globals
int XWindowSize = 1000;
int YWindowSize = 1000; 

// Clip plains
double Near = 0.2;
double Far = 80.0;

//Direction here your eye is located location
double EyeX = 0.0;
double EyeY = 2.0;
double EyeZ = 2.0;

//Where you are looking
double CenterX = 0.0;
double CenterY = 0.0;
double CenterZ = 0.0;

//Up vector for viewing
double UpX = 0.0;
double UpY = 1.0;
double UpZ = 0.0;

void Display(void)
{
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	glutSwapBuffers();
	glFlush();
	control();
}

void reshape(int w, int h)
{
	glViewport(0, 0, (GLsizei) w, (GLsizei) h);

	glMatrixMode(GL_PROJECTION);

	glLoadIdentity();

	glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);

	glMatrixMode(GL_MODELVIEW);
}

int main(int argc, char** argv)
{
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(0,0);
	glutCreateWindow("1D Myocardium");
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutMainLoop();
	return 0;
}

