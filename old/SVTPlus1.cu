#include "hip/hip_runtime.h"
// nvcc SVTPlus.cu -o svtplus -lglut -lm -lGLU -lGL -lmenu -lncurses
//To stop hit "control c" in the window you launched it from.

// Length will be in millimeters
// Time will be in milliseconds
// Mass will be in ???

// Fiber length 100 micrometers or 0.1 millimeters
// Sodium wave speed .5 meters/sec or 0.5 millimeters/millisec

/*
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <curses.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string.h>
using namespace std;
*/

#include <GL/glut.h>
#include <GL/glu.h>
#include <GL/gl.h>
#include <math.h>
#include <stdio.h>
#include "stdio.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <dirent.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <signal.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <time.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
using namespace std;

#define PI 3.141592654

#define STOP_TIME 60000.0
#define DT  0.001

#define NUMBER_OF_NODES 62
#define NUMBER_OF_MUSCLES 132

// Globals
float4 NodePosition[NUMBER_OF_NODES], NodeVelocity[NUMBER_OF_NODES], NodeForce[NUMBER_OF_NODES];
float NodeMass[NUMBER_OF_NODES];
int NodeLinks[NUMBER_OF_NODES][12]; // The nodes that this node is connected to
int NodeMuscles[NUMBER_OF_NODES][12]; // The muscle that connects this node to ther other nodes
int NodeAblatedYesNo[NUMBER_OF_NODES];

// How the muscle will act without contraction.
int MuscleConectionA[NUMBER_OF_MUSCLES];
int MuscleConectionB[NUMBER_OF_MUSCLES];
float MuscleMass[NUMBER_OF_MUSCLES];
float MuscleLength[NUMBER_OF_MUSCLES];
float MuscleRelaxedStrength[NUMBER_OF_MUSCLES];
float MuscleCompresionMultiplier = 10.0;
float MuscleTentionMultiplier = 10.0;
float MuscleCompresionStopFraction[NUMBER_OF_MUSCLES];  // 0.6 is the standard value
float Viscosity = 10.0;
float3 MuscleColor[NUMBER_OF_MUSCLES];

// Muscle contraction parameters
int ContractionOnOff[NUMBER_OF_MUSCLES];
float ContractionTimer[NUMBER_OF_MUSCLES];
float ActionPotentialSpeed[NUMBER_OF_MUSCLES]; //0.5 is a good value.
float ActionPotentialDuration[NUMBER_OF_MUSCLES];
float ContractionDuration[NUMBER_OF_MUSCLES]; // 100.0 is a good value
float RelaxationDuration[NUMBER_OF_MUSCLES]; // 200.0 is a good value
float ContractionStrength[NUMBER_OF_MUSCLES]; // 5.0 is a good value

float BloodPresure = 0.05;

float BeatPeriod = 400.0;

int DrawRate = 1000;

//Globals for setting up the viewing window 
int XWindowSize = 1000;
int YWindowSize = 1000; 
double Near = 0.2;
double Far = 80.0;

/*
double ViewBoxSize = 300.0;

GLdouble Left = -ViewBoxSize;
GLdouble Right = ViewBoxSize;
GLdouble Bottom = -ViewBoxSize;
GLdouble Top = ViewBoxSize;
GLdouble Front = ViewBoxSize;
GLdouble Back = -ViewBoxSize;
*/

//Direction here your eye is located location
double EyeX = 0.0;
double EyeY = 2.0;
double EyeZ = 2.0;

//Where you are looking
double CenterX = 0.0;
double CenterY = 0.0;
double CenterZ = 0.0;

//Up vector for viewing
double UpX = 0.0;
double UpY = 1.0;
double UpZ = 0.0;

// Prototyping functions
void draw_picture();
void generalMuscleForces();
void outwardPresure();
void turnOnNodeMuscles(int);
int contractionForces(float, float);
void dampingForce();
void moveNodes(float, float);
int n_body();
void control();
void mymouse(int, int, int, int);
void Display(void);
void reshape(int, int);
static void signalHandler(int);

int set_initial_conditions()
{	
	int index;
	float dx, dy, dz;
	float sum;
	
	// Node position values for a sphere with 62 nodes//0.5 is a good value.
	NodePosition[0].x = 0.0;
	NodePosition[0].y = 1.0;
	NodePosition[0].z = 0.0;
	NodePosition[NUMBER_OF_NODES-1].x = 0.0;
	NodePosition[NUMBER_OF_NODES-1].y = -1.0;
	NodePosition[NUMBER_OF_NODES-1].z = 0.0;
	
	index = 1;
	for(int i = 1; i < 6; i++)
	{
		for(int j = 0; j < 12; j++)
		{
			if((NUMBER_OF_NODES-1) <= index)
			{
				printf("\nTSU Error: number of nodes is out of bounds\n");
				return(0);
			} 
			NodePosition[index].y = sin(PI/2.0 -i*PI/6.0);
			NodePosition[index].x = cos(PI/2.0 -i*PI/6.0)*cos(j*PI/6.0);
			NodePosition[index].z = cos(PI/2.0 -i*PI/6.0)*sin(j*PI/6.0);
			
			index++;
		}	
	}
	
	// Zeroing out velocity and acceleration
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		NodeVelocity[index].y = 0.0;
		NodeVelocity[index].x = 0.0;
		NodeVelocity[index].z = 0.0;
		
		NodeForce[index].y = 0.0;
		NodeForce[index].x = 0.0;
		NodeForce[index].z = 0.0;
	}
	

	// Below are the edges for the links connecting 62 node sphere.
	// 0: 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12
	
	// 1:  0  12 2  13		13: 1  24 14 25		25: 13 36 26 37		37: 25 48 38 49		49: 37 60 50 61
	// 2:  0  1  3  14		14: 2  13 15 26		26: 14 25 27 38		38: 26 37 39 50		50: 38 49 51 61
	// 3:  0  2  4  15		15: 3  14 16 27		27: 15 26 27 39		39: 27 38 40 51		51: 39 50 52 61
	// 4:  0  3  5  16		16: 4  15 17 28		28: 16 27 27 40		40: 28 39 41 52		52: 40 51 53 61
	// 5:  0  4  6  17		17: 5  16 18 29		29: 17 28 27 41		41: 29 40 42 53		53: 41 52 54 61
	// 6:  0  5  7  18		18: 6  17 19 30		30: 18 29 27 42		42: 30 41 43 54		54: 42 53 55 61
	// 7:  0  6  8  19		19: 7  18 20 31		31: 19 30 27 43		43: 31 41 44 55		55: 43 54 56 61
	// 8:  0  7  9  20		20: 8  19 21 32		32: 20 31 27 44		44: 32 43 45 56		56: 44 55 57 61
	// 9:  0  8  10 21		21: 9  20 22 33		33: 21 32 27 45		45: 33 44 46 57		57: 45 56 58 61
	// 10: 0  9  11 22		22: 10 21 23 34		34: 22 33 27 46		46: 34 45 47 58		58: 46 57 59 61
	// 11: 0  10 12 23		23: 11 22 24 35		35: 23 34 27 47		47: 35 46 48 59		59: 47 58 60 61
	// 12: 0  11 1  24		24: 12 23 13 36		36: 24 35 25 48		48: 36 47 37 60		60: 48 59 49 61
	
	// 61: 49 50 51 52 53 54 55 56 57 58 59 60
	
	// Setting the nodes to -1 so you can tell the nodes that where not used.
	// The first and the last nodes had 12 links so I made them all have 12.
	// The rest only had 4 so you may want to revisit this.
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		for(int j = 0; j < 12; j++)
		{
			NodeLinks[i][j] =  -1;
			NodeMuscles[i][j] = -1;
		}	
	}
	
	// Setting all nodes as not ablated
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		NodeAblatedYesNo[i] = 0;
	}
	
	// Setting edges for the 0th node.
	for(int i = 0; i < 12; i++)
	{
		NodeLinks[0][i] =  i + 1;
	}
	
	// Setting the edges that are connected to the 0th node
	for(int j = 0; j < 4; j++)
	{
		for(int i = 1; i < 13; i++)
		{
			// Connect to node above
			if(j == 0)
			{
				NodeLinks[i][j] =  0;
			}
			
			// Connect to the node to the left
			if(j == 1)
			{
				NodeLinks[i][j] =  (i+10)%12 + 1;
			}
			
			// Connect to the node to the right
			if(j == 2)
			{
				NodeLinks[i][j] =  (i+12)%12 + 1;
			}
			
			// Connect to the node below
			if(j == 3)
			{
				NodeLinks[i][j] =  i + 12;
			}
		}
		
		// Setting the middle 3 sections
		for(int k = 0; k <= 3*12; k += 12)
		{
			for(int i = 13 + k; i < 25 + k; i++)
			{
				// Connect to node above
				if(j == 0)
				{
					NodeLinks[i][j] =  i - 12;
				}
				
				// Connect to the node to the left
				if(j == 1)
				{
					NodeLinks[i][j] =  (i+10)%12 + 13 + k;
				}
				
				// Connect to the node to the right
				if(j == 2)
				{
					NodeLinks[i][j] =  (i+12)%12 + 13 + k;
				}
				
				// Connect to the node below
				if(j == 3)
				{
					NodeLinks[i][j] =  i + 12;
				}
			}
		}
		
		// Setting the edges that are linked to the 61st node
		for(int i = 49; i < 61; i++)
		{
			// Connect to node above
			if(j == 0)
			{
				NodeLinks[i][j] =  i - 12;
			}
			
			// Connect to the node to the left
			if(j == 1)
			{
				NodeLinks[i][j] =  (i+10)%12 + 49;
			}
			
			// Connect to the node to the right
			if(j == 2)
			{
				NodeLinks[i][j] =  (i+12)%12 + 49;
			}
			
			// Connect to the node below
			if(j == 3)
			{
				NodeLinks[i][j] =  NUMBER_OF_NODES - 1;
			}
		}
		
		// Setting the 61st node.
		for(int i = 0; i < 12; i++)
		{
			NodeLinks[61][i] =  i + 49;
		}
	}
	
	//Setting the ends of the muscles
	index = 0;
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		for(int j = 0; j < 12; j++)
		{
			if(NodeLinks[i][j] != -1)
			{
				if(i < NodeLinks[i][j])
				{
					if(NUMBER_OF_MUSCLES <= index)
					{
						printf("\nTSU Error: number of edges is out of bounds\n");
						return(0);
					} 
					MuscleConectionA[index] = i;
					MuscleConectionB[index] = NodeLinks[i][j];
					index++;
				}
			}
		}
	}
	
	// Setting the node muscles. Each node will have a list of nodes they are attached to (NodeLinks[][]) and the muscle that attaches it to that node (NodeMuscles[][]).
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		for(int j = 0; j < 12; j++)
		{
			if(NodeLinks[i][j] != -1)
			{
				for(int k = 0; k < NUMBER_OF_MUSCLES; k++)
				{
					if((MuscleConectionA[k] == i && MuscleConectionB[k] == NodeLinks[i][j]) || (MuscleConectionA[k] == NodeLinks[i][j] && MuscleConectionB[k] == i))
					{
						NodeMuscles[i][j] = k;
					}
				}
			}
		}
	}
	
	// Setting other parameters
	for(int i = 0; i < NUMBER_OF_MUSCLES; i++)
	{	
		MuscleMass[i] = 1.0;
		dx = NodePosition[MuscleConectionA[i]].x - NodePosition[MuscleConectionB[i]].x;
		dy = NodePosition[MuscleConectionA[i]].y - NodePosition[MuscleConectionB[i]].y;
		dz = NodePosition[MuscleConectionA[i]].z - NodePosition[MuscleConectionB[i]].z;
		MuscleLength[i] = sqrt(dx*dx + dy*dy + dz*dz);;
		MuscleRelaxedStrength[i] = 0.1;
		MuscleCompresionStopFraction[i] = 0.6;
		ContractionOnOff[i] = 0;
		ContractionTimer[i] = 0.0;
		ActionPotentialSpeed[i] = 0.02; // 0.2
		ActionPotentialDuration[i] = MuscleLength[i]/ActionPotentialSpeed[i];
		ContractionDuration[i] = 100.0;  // 100.0
		RelaxationDuration[i] = 200.0;  // 200.0
		ContractionStrength[i] = 0.1;
		MuscleColor[i].x = 1.0;
		MuscleColor[i].y = 0.0;
		MuscleColor[i].z = 0.0;
	}
	
	//Nodes to ablate
	for(int i = 35; i < 50; i++)
	{	
		//NodeAblatedYesNo[i] = 1;
	}
	
	NodeAblatedYesNo[1] = 1;
	NodeAblatedYesNo[13] = 1;
	NodeAblatedYesNo[25] = 1;
	NodeAblatedYesNo[37] = 1;
	NodeAblatedYesNo[49] = 1;
	
	NodeAblatedYesNo[2] = 1;
	NodeAblatedYesNo[3] = 1;
	NodeAblatedYesNo[4] = 1;
	NodeAblatedYesNo[5] = 1;
	NodeAblatedYesNo[6] = 1;
	NodeAblatedYesNo[7] = 1;
	NodeAblatedYesNo[8] = 1;
	
	NodeAblatedYesNo[9] = 1;
	NodeAblatedYesNo[10] = 1;
	NodeAblatedYesNo[11] = 1;
	NodeAblatedYesNo[14] = 1;
	NodeAblatedYesNo[17] = 1;
	
	// Setting the node masses
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		sum = 0.0;
		for(int j = 0; j < 12; j++)
		{
			if(NodeMuscles[i][j] != -1)
			{
				sum += MuscleMass[NodeMuscles[i][j]];
			}
		}
		NodeMass[i] = sum/2.0;
		printf("\nNodeMass[%d] = %f", i, NodeMass[i]);
	}

	return(1);
}

void draw_picture()
{
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	
	glColor3d(1.0,1.0,0.0);
	glPushMatrix();
	glTranslatef(NodePosition[0].x, NodePosition[0].y, NodePosition[0].z);
	glutSolidSphere(0.03,20,20);
	glPopMatrix();
	
	// Drawing nodes
	for(int i = 1; i < NUMBER_OF_NODES; i++)
	{
		if(NodeAblatedYesNo[i] == 0)
		{
			glColor3d(0.0,1.0,0.0);
		}
		else
		{
			glColor3d(1.0,1.0,1.0);
		}
		glPushMatrix();
		glTranslatef(NodePosition[i].x, NodePosition[i].y, NodePosition[i].z);
		glutSolidSphere(0.01,20,20);
		glPopMatrix();	
	}
	
	// Drawing muscles
	glColor3d(1.0,0.0,0.0);
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		for(int j = 0; j < 12; j++)
		{
			if(NodeLinks[i][j] != -1)
			{
				glColor3d(MuscleColor[NodeMuscles[i][j]].x, MuscleColor[NodeMuscles[i][j]].y, MuscleColor[NodeMuscles[i][j]].z);
				//glLineWidth(1.0/(Px[i]-Px[NodeLinks[i][j]]));
				glBegin(GL_LINES);
					glVertex3f(NodePosition[i].x, NodePosition[i].y, NodePosition[i].z);
					glVertex3f(NodePosition[NodeLinks[i][j]].x, NodePosition[NodeLinks[i][j]].y, NodePosition[NodeLinks[i][j]].z);
				glEnd();
			}
			
		}	
	}
	glutSwapBuffers();
}

void moveView()
{
	cout << "\nEnter the desired draw rate: ";
	cin >> DrawRate;
}

void generalMuscleForces()
{
	float f; 
	float dx, dy, dz, d;
	int muscleNumber, nodeNumber;
	
	// Getting forces on the nodes from the muscle fiber without contraction	
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		for(int j = 0; j < 12; j++)
		{
			muscleNumber = NodeMuscles[i][j];
			nodeNumber = NodeLinks[i][j];
			if(nodeNumber != -1)
			{
				dx = NodePosition[nodeNumber].x - NodePosition[i].x;
				dy = NodePosition[nodeNumber].y - NodePosition[i].y;
				dz = NodePosition[nodeNumber].z - NodePosition[i].z;
				d  = sqrt(dx*dx + dy*dy + dz*dz);
				if(d < MuscleCompresionStopFraction[muscleNumber]*MuscleLength[muscleNumber])
				{
					f  = MuscleRelaxedStrength[muscleNumber]*MuscleCompresionMultiplier*(d - MuscleLength[muscleNumber]);
				}
				else if(d < MuscleLength[muscleNumber])
				{
					f  = MuscleRelaxedStrength[muscleNumber]*(d - MuscleLength[muscleNumber]);
				}
				else
				{
					f  = MuscleRelaxedStrength[muscleNumber]*MuscleTentionMultiplier*(d - MuscleLength[muscleNumber]);
				}
				NodeForce[i].x  += f*dx/d;
				NodeForce[i].y  += f*dy/d;
				NodeForce[i].z  += f*dz/d;
			}
		}
	}
}

void outwardPresure()
{
	float f; 
	float dx, dy, dz, d;
	float4 centerOfMass;
	
	
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		 centerOfMass.x += NodePosition[i].x*NodeMass[i];
		 centerOfMass.y += NodePosition[i].y*NodeMass[i];
		 centerOfMass.z += NodePosition[i].z*NodeMass[i];
		 centerOfMass.w += NodeMass[i];
	}
	
	centerOfMass.x /= centerOfMass.w;
	centerOfMass.y /= centerOfMass.w;
	centerOfMass.z /= centerOfMass.w;
		 
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		 NodePosition[i].x -= centerOfMass.x;
		 NodePosition[i].y -= centerOfMass.y;
		 NodePosition[i].z -= centerOfMass.z;
	}
	
	// Getting forces on the nodes from the presure of the blood pushing out	
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		dx = 0.0 - NodePosition[i].x;
		dy = 0.0 - NodePosition[i].y;
		dz = 0.0 - NodePosition[i].z;
		d  = sqrt(dx*dx + dy*dy + dz*dz);
		
		f  = -BloodPresure;
		
		NodeForce[i].x  += f*dx/d;
		NodeForce[i].y  += f*dy/d;
		NodeForce[i].z  += f*dz/d;
	}
}

void turnOnNodeMuscles(int index)
{
	for(int j = 0; j < 12; j++)
	{
		if((NodeLinks[index][j] != -1) && (ContractionOnOff[NodeMuscles[index][j]] == 0))
		{
			ContractionOnOff[NodeMuscles[index][j]] = 1;
			ContractionTimer[NodeMuscles[index][j]] = 0.0;
		}
	}
}

int contractionForces(float dt, float time)
{
	float dx, dy, dz, d;
	int muscleNumber, nodeNumber;
	
	// Getting forces for the muscle fiber contraction
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		for(int j = 0; j < 12; j++)
		{
			muscleNumber = NodeMuscles[i][j];
			nodeNumber = NodeLinks[i][j];
			if(nodeNumber != -1)
			{	
				if(ContractionOnOff[muscleNumber] == 1)
				{
					if((ActionPotentialDuration[muscleNumber] - dt < ContractionTimer[muscleNumber]) && (ContractionTimer[muscleNumber] < ActionPotentialDuration[muscleNumber] + dt))
					{
						if(NodeAblatedYesNo[nodeNumber] == 0)
						{
							turnOnNodeMuscles(nodeNumber);
						}
					}
					
					if(ContractionTimer[muscleNumber] < ActionPotentialDuration[muscleNumber])
					{
						MuscleColor[muscleNumber].x = 1.0;
						MuscleColor[muscleNumber].y = 1.0;
						MuscleColor[muscleNumber].z = 1.0;
					}
					else
					{
						MuscleColor[muscleNumber].x = 1.0;
						MuscleColor[muscleNumber].y = 0.0;
						MuscleColor[muscleNumber].z = 0.0;
					}
					
					if(ContractionTimer[muscleNumber] < ContractionDuration[muscleNumber])
					{
						dx = NodePosition[nodeNumber].x - NodePosition[i].x;
						dy = NodePosition[nodeNumber].y - NodePosition[i].y;
						dz = NodePosition[nodeNumber].z - NodePosition[i].z;
						d  = sqrt(dx*dx + dy*dy + dz*dz);
						
						NodeForce[i].x   += ContractionStrength[muscleNumber]*dx/d;
						NodeForce[i].y   += ContractionStrength[muscleNumber]*dy/d;
						NodeForce[i].z   += ContractionStrength[muscleNumber]*dz/d;
					
						ContractionTimer[muscleNumber] += dt;
					}
					else if(ContractionTimer[muscleNumber] < ContractionDuration[muscleNumber] + RelaxationDuration[muscleNumber])
					{
						NodeForce[i].x   += 0.0;
						NodeForce[i].y   += 0.0;
						NodeForce[i].z   += 0.0;
						
						ContractionTimer[muscleNumber] += dt;
					}
					else
					{
						ContractionOnOff[muscleNumber] = 0;
						ContractionTimer[muscleNumber] = 0.0;
					}
				}
			}
		}
	}
	return(1);
}

void dampingForce()
{
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{	
		NodeForce[i].x   += -Viscosity*NodeVelocity[i].x;
		NodeForce[i].y   += -Viscosity*NodeVelocity[i].y;
		NodeForce[i].z   += -Viscosity*NodeVelocity[i].z;
	}
}

void moveNodes(float dt, float time)  // LeapFrog
{
	// Moving the system forward in time with leap-frog.
	for(int i = 0; i < NUMBER_OF_NODES; i++)
	{
		if(time == 0.0)
		{
			NodeVelocity[i].x += (NodeForce[i].x/MuscleMass[i])*0.5*dt;
			NodeVelocity[i].y += (NodeForce[i].y/MuscleMass[i])*0.5*dt;
			NodeVelocity[i].z += (NodeForce[i].z/MuscleMass[i])*0.5*dt;
		}
		else
		{
			NodeVelocity[i].x += (NodeForce[i].x/MuscleMass[i])*dt;
			NodeVelocity[i].y += (NodeForce[i].y/MuscleMass[i])*dt;
			NodeVelocity[i].z += (NodeForce[i].z/MuscleMass[i])*dt;
		}

		NodePosition[i].x += NodeVelocity[i].x*dt;
		NodePosition[i].y += NodeVelocity[i].y*dt;
		NodePosition[i].z += NodeVelocity[i].z*dt;
	}
}

int n_body()
{
	int   tdraw = 0; 
	double time = 0.0;
	float beatTimer = 0.0;
	
	//mousemask(ALL_MOUSE_EVENTS | REPORT_MOUSE_POSITION, NULL);
	//MEVENT event;
	
	//int mouse;
	
	while(time < STOP_TIME)
	{
		if(BeatPeriod <= beatTimer)
		{
			turnOnNodeMuscles(0);
			beatTimer = 0.0;
		}
		else beatTimer += DT;
		
		// Zeroing out the nodal forces.
		for(int i = 0; i < NUMBER_OF_NODES; i++)
		{
			NodeForce[i].x   = 0.0;
			NodeForce[i].y   = 0.0;
			NodeForce[i].z   = 0.0;
		}
		
		generalMuscleForces();
		
		contractionForces(DT, time);
		
		outwardPresure();
		
		dampingForce();
		
		moveNodes(DT, time);

		if(tdraw == DrawRate) 
		{
			draw_picture();
			tdraw = 0;
			printf("\n Time = %f", time);
		}
		else tdraw++;
		
		time += DT;
	}
	return(1);
}

void control()
{	
	//int    tdraw = 0;
	//float  time = 0.0;
	
	struct sigaction sa;
	
	sa.sa_handler = signalHandler;
	sigemptyset(&sa.sa_mask);
	sa.sa_flags = SA_RESTART; // Restart functions if interrupted by handler
	if (sigaction(SIGINT, &sa, NULL) == -1)
	{
		printf("\nTSU Error: sigaction error\n");
	}

	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);

	set_initial_conditions();
	
	draw_picture();
	
	if(n_body() == 1) printf("\n N-body success \n");
	
	printf("\n DONE \n");
	while(1);
}

void mymouse(int button, int state, int x, int y)
{	
	if(state == GLUT_DOWN)
	{
		if(button == GLUT_LEFT_BUTTON)
		{
			printf("\n  x = %d", x);
		}
		else
		{
			printf("\n  y = %d", y);
		}
	}
}

void Display(void)
{
	gluLookAt(EyeX, EyeY, EyeZ, CenterX, CenterY, CenterZ, UpX, UpY, UpZ);
	glClear(GL_COLOR_BUFFER_BIT);
	glClear(GL_DEPTH_BUFFER_BIT);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	glutSwapBuffers();
	glFlush();
	glutMouseFunc(mymouse);
	control();
}

void reshape(int w, int h)
{
	glViewport(0, 0, (GLsizei) w, (GLsizei) h);

	glMatrixMode(GL_PROJECTION);

	glLoadIdentity();

	glFrustum(-0.2, 0.2, -0.2, 0.2, Near, Far);

	glMatrixMode(GL_MODELVIEW);
}

static void signalHandler(int signum)
{
	int command;
    
	cout << "\n\n******************************************************" << endl;
	cout << "Enter:666 to kill the run." << endl;
	cout << "Enter:1 to change the draw rate." << endl;
	cout << "Enter:2 Move view" << endl;
	cout << "Enter:3 to continue the run." << endl;
	cout << "******************************************************\n\nCommand: ";
    
	cin >> command;
    
	if(command == 666)
	{
		exit(0);
	}
	else if(command == 1)
	{
		cout << "\nEnter the desired draw rate: ";
		cin >> DrawRate;
		cout << "\nDrawRate: " << DrawRate << endl;
	}
	else if (command == 2)
	{
		moveView();
	}
	else if (command == 3)
	{
		cout << "\nRun continued." << endl;
	}
	else
	{
		cout <<"\n\n Invalid Command\n" << endl;
	}
}

int main(int argc, char** argv)
{
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(0,0);
	glutCreateWindow("SVT Plus");
	GLfloat light_position[] = {1.0, 1.0, 1.0, 0.0};
	GLfloat light_ambient[]  = {0.0, 0.0, 0.0, 1.0};
	GLfloat light_diffuse[]  = {1.0, 1.0, 1.0, 1.0};
	GLfloat light_specular[] = {1.0, 1.0, 1.0, 1.0};
	GLfloat lmodel_ambient[] = {0.2, 0.2, 0.2, 1.0};
	GLfloat mat_specular[]   = {1.0, 1.0, 1.0, 1.0};
	GLfloat mat_shininess[]  = {10.0};
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glShadeModel(GL_SMOOTH);
	glColorMaterial(GL_FRONT, GL_AMBIENT_AND_DIFFUSE);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
	glLightModelfv(GL_LIGHT_MODEL_AMBIENT, lmodel_ambient);
	glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
	glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable(GL_COLOR_MATERIAL);
	glEnable(GL_DEPTH_TEST);
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutMainLoop();
	return 0;
}

